#include "hip/hip_runtime.h"
/* move extern int Lidx_global and extern int octant_global to arr.h  2020.12.29*/
/* fix typo in cal_dis_O() 2020.12.30 */
/* add timing in array_add_ylm() 2020.12.31 */
/* add r_max debug in array_add_ylm() 2020.12.31 */
/* add auto renaming output file in write_array() 2021.01.03 */
/* add iter_test for testing the code serval times 2021.01.07*/
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include"macros.h"
#include"text_io.h"
#include"main.h"
#include"arr.h"
#include"ylm.h"
#include"cuapi.h"
#include"run_halo.h"
#define RESTART_FILENAME_R "RESTART_R"
#define RESTART_FILENAME_I "RESTART_I"

char buff_r[128], buff_i[128];
void run_single_lidx(void);

void run(void)
{
    run_single_lidx();
}

void run_single_lidx(void)
{
    float time_temp;
    
//    solve_init();
//    if(rank==0)
//        gen_r(r_1,r);
//    MPI_Barrier(MPI_COMM_WORLD);
//    MPI_Bcast(r, rsiz, MPI_DOUBLE, 0, MPI_COMM_WORLD);
//    MPI_Bcast(r_1, rsiz, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    //printf("r[0]=%e 0\n",r[0]);//debug
	
    int l_max=0;
    int l_init=0;
    int counter=1;
    bool cuda_error_flag;
    
/// call r_max ///
#ifdef OCTANT_DECOMPOSE
    cal_r_max_O();
#else
    cal_r_max();
#endif
#ifdef DEBUG
    printf("r_max found! r_max = %.4e .\n", r_max);
#endif
    strcpy(buff_r, file_r);
    strcpy(buff_i, file_i);

    array_init();
    if (restart_flag==1)
    {
        read_array(true, restart_id, RESTART_FILENAME_R, RESTART_FILENAME_I);
        printf("Restart from Lidx=%d .\n", restart_id);
    }
    else
        restart_id = 0;

#ifdef DEBUG
    printf("CUDA halo reconstruction starts...\n");
#endif

    hipEvent_t start, stop, eigen_copy_start, eigen_copy_stop, amplitude_copy_start, amplitude_copy_stop, lidx_start, lidx_stop;
    cuda_error_flag = CUDA_CHECK_ERROR(hipSetDevice(0));
    cuda_error_flag = CUDA_CHECK_ERROR(hipEventCreate(&start));
    cuda_error_flag = CUDA_CHECK_ERROR(hipEventCreate(&stop));
    cuda_error_flag = CUDA_CHECK_ERROR(hipEventCreate(&eigen_copy_start));
    cuda_error_flag = CUDA_CHECK_ERROR(hipEventCreate(&eigen_copy_stop));
    cuda_error_flag = CUDA_CHECK_ERROR(hipEventCreate(&amplitude_copy_start));
    cuda_error_flag = CUDA_CHECK_ERROR(hipEventCreate(&amplitude_copy_stop));
    cuda_error_flag = CUDA_CHECK_ERROR(hipEventCreate(&lidx_start));
    cuda_error_flag = CUDA_CHECK_ERROR(hipEventCreate(&lidx_stop));
    cuda_error_flag = CUDA_CHECK_ERROR(hipEventRecord(start,0));
    	
// add data dump and restart 2021.01.04
    for(int lidx=restart_id;lidx<lnod;lidx++)
    {
        Lidx_global = lidx;
        cuda_error_flag = CUDA_CHECK_ERROR(hipEventRecord(lidx_start,0));
        cuda_error_flag = CUDA_CHECK_ERROR(hipEventRecord(eigen_copy_start,0));
        
        /*** load eigenstates and broadcast to every nodes***/
        load_eigenstates(&l_init,&l_max,&eigen_num,"egn",Lidx_global);
#ifdef DEBUG
        printf("Eigen states loading completed!\n");
#endif
        printf("Total number of qunatum number n is %d .\n", eigen_num);
        cuda_error_flag = CUDA_CHECK_ERROR(hipEventRecord(eigen_copy_stop,0));
        cuda_error_flag = CUDA_CHECK_ERROR(hipEventSynchronize(eigen_copy_stop));
        cuda_error_flag = CUDA_CHECK_ERROR(hipEventElapsedTime(&time_temp, eigen_copy_start, eigen_copy_stop));
        printf("Total time for copying eigen states from l_init=%d to l_max=%d is %.4e s.\n", l_init, l_max, time_temp/1000.);
//  ignoring all operations if l_max<l_init 2020.12.29
        if (l_max<l_init)
        {
            printf("l_init = %d @ Lidx = %d will be ignored since l_max < l_init !\n", l_init, Lidx_global);
            continue;
        }
#ifdef DEBUG
        printf("Start kernel preparation...\n");
#endif
        /*** generate upper bound ***/
//        gen_ubound(rv[0][0] * 0., l_max,l_init);
        ylm_init(l_init,l_max);
#ifdef DEBUG
        printf("Memory allocation for spherical harmonics completed.\n");
#endif

        cuda_error_flag = CUDA_CHECK_ERROR(hipEventRecord(amplitude_copy_start,0));
        /*** read amplitudes and Bcast ***/
        load_ylm(l_init,l_max,"amp_only",Lidx_global);
        /*** generate density in simulation box ***/
#ifdef DEBUG
        printf("Amplitudes of spherical harmonics loading completed.\n");
#endif
        cuda_error_flag = CUDA_CHECK_ERROR(hipEventRecord(amplitude_copy_stop,0));
        cuda_error_flag = CUDA_CHECK_ERROR(hipEventSynchronize(amplitude_copy_stop));
        cuda_error_flag = CUDA_CHECK_ERROR(hipEventElapsedTime(&time_temp, amplitude_copy_start, amplitude_copy_stop));
        printf("Total time for copying  amplitudes from l_init=%d to l_max=%d is %.4e s.\n", l_init, l_max, time_temp/1000.);

        array_add_ylm(l_max,l_init,true);

        cuda_error_flag = CUDA_CHECK_ERROR(hipDeviceSynchronize());
        ylm_fin(l_init,l_max);
        free_egn(l_max,l_init);

        if (dump_flag==1&&(counter%dump_interval)==0)
        {
            printf("===============================================\n");
            printf("Data dumped at Lidx=%d.\n", Lidx_global);
            printf("===============================================\n");
            write_array(true,&Lidx_global,file_r,file_i);
            strcpy(file_r, buff_r);
            strcpy(file_i, buff_i);
        }

        cuda_error_flag = CUDA_CHECK_ERROR(hipEventRecord(lidx_stop,0));
        cuda_error_flag = CUDA_CHECK_ERROR(hipEventSynchronize(lidx_stop));
        cuda_error_flag = CUDA_CHECK_ERROR(hipEventElapsedTime(&time_temp, lidx_start, lidx_stop));
        printf("Total time for running through l_init=%d to l_max=%d takes %.4e s.\n",l_init, l_max, time_temp/1000.);

        counter ++;
    }
    printf("Output 3D wave function...\n");
    write_array(false,&Lidx_global,file_r,file_i);

    cuda_error_flag = CUDA_CHECK_ERROR(hipEventRecord(stop,0));
    cuda_error_flag = CUDA_CHECK_ERROR(hipEventSynchronize(stop));
    cuda_error_flag = CUDA_CHECK_ERROR(hipEventElapsedTime(&time_temp, start, stop));
    time_temp /= 1000.;
    int total_hr = (int)(time_temp/3600);
    int total_min = (int)((time_temp-3600*total_hr)/60);
    double total_sec = time_temp - 3600.*total_hr - 60.*total_min;
    printf("Total computation time for reconstructing halo is %d hr %d min %.2f s .\n", total_hr, total_min, total_sec);

    array_fin();
    hipDeviceReset();
    return;
}
