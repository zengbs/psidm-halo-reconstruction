#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<string.h>
#include<hip/hip_runtime.h>
#include"macros.h"
#include"main.h"
#include"ylm.h"
#include"arr.h"
#include"cuapi.h"

// for nan debugging
#define nan_index_x 272 //0
#define nan_index_y 220 //0
#define nan_index_z 0 //1


int lnodidx = Lidx_global;

__device__ void cal_index_r(int rsiz, float r, float dr, int *index_r, float *frac_r)
{
    float rr = r/dr;
    int i = (int)floor(rr);
    if(i >= rsiz)
        i = rsiz - 1;
    if(i <= 0)
        i = 1;
    *frac_r = rr - i;
    *index_r = i;
    return;
}

#ifdef OCTANT_DECOMPOSE
__global__ void cal_and_expand_ylm_O(int rsiz, int nx, int ny, int nz, int octant, float dx, float dy, float dz, float xcen, float ycen, float zcen, float dr, float r_max, float *arr_r, float *arr_i, int *rfunc_num, float ***rfunc, float ***amplitude_r, float ***amplitude_i, int l_max,int l_init)
#else
__global__ void cal_and_expand_ylm(int rsiz, int nx, int ny, int nz, float dx, float dy, float dz, float xcen, float ycen, float zcen, float dr, float r_max, float *arr_r, float* arr_i, int *rfunc_num, float ***rfunc, float ***amplitude_r , float ***amplitude_i, int l_max, int l_init)
#endif
{
//if (blockIdx.x==0&&blockIdx.y==0&&blockIdx.z==0)
//    printf("%d, %s\n", __LINE__, __FUNCTION__);
    int idx_x = threadIdx.x + blockIdx.x*blockDim.x;
    int idx_y = threadIdx.y + blockIdx.y*blockDim.y;
    int idx_z = threadIdx.z + blockIdx.z*blockDim.z;

//    extern __shared__ float rfunc_interp[];

    if (idx_x<nx&&idx_y<ny&&idx_z<nz)
    {
        float x, y, z, r;
#ifdef OCTANT_DECOMPOSE
        cal_dis_x_O(idx_x, nx, octant, dx, xcen, &x);
        cal_dis_y_O(idx_y, ny, octant, dy, ycen, &y);
        cal_dis_z_O(idx_z, nz, octant, dz, zcen, &z);
#else
        cal_dis_x(idx_x, dx, xcen, &x);
        cal_dis_y(idx_y, dy, ycen, &y);
        cal_dis_z(idx_z, dz, zcen, &z);
#endif
        cal_dis_r(&x, &y, &z, &r);        
//#ifdef DEBUG
//        if (idx_x==nan_index_x&&idx_y==nan_index_y&&idx_z==nan_index_z)
//        if (idx_x==(int)(nx/2)&&idx_y==(int)(ny/2)&&idx_z==(int)(nz/2))
//        {
//            printf("Kernel executing...\n");
//            printf("Calculating distance r completed.\n");
//            printf("x = %.4e , y = %.4e , z= %.4e , r = %.4e , r_max = %.4e .\n", x, y, z, r, r_max);
//              printf("index_x = %d , index_y = %d, index_z = %d ; x_cen = %.6e , y_cen = %.6e z_cen = %.6e ; r = %.8e, r_max = %.8e .\n", idx_x, idx_y, idx_z, xcen, ycen, zcen, r, r_max);
//        }
//#endif
        if (r<r_max)
        {
            int n, l, m, ll;
            int index_r, index_site;
            float frac_r, rfunc_interp;
            float fact_1, fact_2 = 0., oldfact = 0.;  // for ylm computation
            float pmm_new, pmm_old, phr_new, phr_old, phi_new, phi_old;
            float array_pos[2];
            double pml, pml_lm1, temp;
            double ylmr[2];

            cal_index_r(rsiz, r, dr, &index_r, &frac_r);
            index_site = get_pos(idx_x, idx_y, idx_z, nx, ny);
//            index_thread = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
//#ifdef DEBUG
//            if (idx_x==(int)(nx/2)&&idx_y==(int)(ny/2)&&idx_z==(int)(nz/2))
//                printf("index_site = %d ; index_r = %d ; frac_r = %.4e .\n", index_site, index_r, frac_r);
//#endif
            array_pos[0] = (float)0.;
            array_pos[1] = (float)0.;

            x /= r;
            y /= r;
            z /= r;
            fact_1 = -(float)1.;
//            if (idx_x==nan_index_x&&idx_y==nan_index_y&&idx_z==nan_index_z)
//                printf("x = %.8e\t y = %.8e\t z = %.8e\n", x, y, z);

//            int counter = 0;
//            int shift = ln_total_count*index_thread;
//            for (l=0; l<=l_max-l_init;l++)
//            {
//                for (n=0; n<rfunc[l]; n++)
//                    *(rfunc_interp+shift+counter+n) = (float)(rfunc[ll][n][index_r-1])*(1-frac_r) + (float)(rfunc[ll][n][index_r])*(frac_r);
//                counter += rfunc[l];
//            }

            for (m=0; m<=l_max; m++)
            {
                if (m==0)
                {
                    phr_new = (float)1.;
                    phi_new = (float)0.;
                    pmm_new = (float)1.;
                    pmm_old = (float)1.;
                }
                else
                {
                    if (m==1)
                    {
                        phr_new = x;
                        phi_new = y;
                        phr_old = phr_new;
                        phi_old = phi_new;
                    }
                    else
                    {
                        phr_new = phr_old*x - phi_old*y;
                        phi_new = phi_old*x + phr_old*y;
                        phr_old = phr_new;
                        phi_old = phi_new;
                    }
                    pmm_new = fact_1 * pmm_old / (fact_1+(float)1.);
//                    if (idx_x==nan_index_x&&idx_y==nan_index_y&&idx_z==nan_index_z)
//                        printf("pmm_new = %.4e\n", pmm_new);
                    pmm_old = pmm_new;
                }
                fact_1 += (float)2.;
        
                pmm_new = sqrtf(((float)2.*m+(float)1.)*pmm_new/((float)4.*(float)M_PI));
//                if (idx_x==nan_index_x&&idx_y==nan_index_y&&idx_z==nan_index_z)
//                   printf("pmm_new = %.4e\n", pmm_new);
        
                for(l = m;l <= l_max; l++)
                {
                    if(l > m)
                    {
                        fact_2 = sqrtf(((float)4.*l*l-(float)1.)/(l*l-m*m));
//                        if (idx_x==nan_index_x&&idx_y==nan_index_y&&idx_z==nan_index_z)
//                           printf("fact_2 = %.4e\n", fact_2);
                    }
                    if(l == m)
                        pml = pmm_new;
                    else if(l == m+(float)1.)
                    {
                        pml_lm1 = pml;
                        pml = fact_2*z*pml;
//#ifdef DEBUG
//                        if (idx_x==0 && idx_y==0 && idx_z==0)
//                        {
//                            if (m==0&&l==1)
//                                printf("fact_2 = %.6e ; z = %.6e ; pml_lm1 = %.6e ; pml = %.6e .\n", fact_2, z, pml_lm1, fact_2*z*pml_lm1);
//                        }
//#endif
                    }
                    else
                    {
                        temp = pml;
                        pml = fact_2*(z*pml-pml_lm1/oldfact);
//                        if (idx_x==nan_index_x&&idx_y==nan_index_y&&idx_z==nan_index_z)
//                           printf("pml = %.4e\n", pml);
                        pml_lm1 = temp;
                    }
                    oldfact = fact_2;
                    if(m!=0)
                    {
                        ylmr[0] = pml * phr_new * sqrtf((float)2.); // ylmr[0] = ylm[l][m]
                        ylmr[1] = pml * phi_new * sqrtf((float)2.); // ylmr[1] = ylmr[l][-m+2*l+1]
                        // to avoid nan
//                        if (ylmr[0]!=ylmr[0])
//                            ylmr[0] = (float)(0.);
//                        if (ylmr[1]!=ylmr[1])
//                            ylmr[1] = (float)(0.);
                        //
                        ll = l-l_init;
                        if (ll>=0)
                        {
                            for (n=0; n<rfunc_num[ll]; n++)
                            {
                                rfunc_interp = rfunc[ll][n][index_r-1]*((float)1.-frac_r) + rfunc[ll][n][index_r]*frac_r;
                                array_pos[0] += (amplitude_r[ll][n][m]*(float)(ylmr[0]) + amplitude_r[ll][n][-m+2*l+1]*(float)(ylmr[1]))*rfunc_interp;
                                array_pos[1] += (amplitude_i[ll][n][m]*(float)(ylmr[0]) + amplitude_i[ll][n][-m+2*l+1]*(float)(ylmr[1]))*rfunc_interp;
//#ifdef DEBUG
//                                if (idx_x==nan_index_x && idx_y==nan_index_y && idx_z==nan_index_z)
//                                {
//                                    printf("ylm[%d][%d] = %.6e .\n", l, m, ylmr[0] );
//                                    printf("ylm[%d][%d] = %.6e .\n", l, -m+2*l+1, ylmr[1] );
//                                    printf("amplitude_r[%d][%d][%d] = %.6e .\n", l, n ,m, amplitude_r[ll][n][m]);
//                                    printf("amplitude_i[%d][%d][%d] = %.6e .\n", l, n ,m, amplitude_i[ll][n][m]);
//                                    printf("rfunc_interp = %.6e .\n", rfunc_interp);
//                                }
//#endif
                            }
                        }
                    }
                    else
                    {
                        ylmr[0] = pml * phr_new;
                        // to avoid nan
//                        if (ylmr[0]!=ylmr[0])
//                            ylmr[0] = (float)(0.);
                        //
                        ll = l-l_init;
                        if (ll>=0)
                        {
                            for (n=0; n<rfunc_num[ll]; n++)
                            {
                                rfunc_interp = rfunc[ll][n][index_r-1]*((float)1.-frac_r) + rfunc[ll][n][index_r]*frac_r;
                                array_pos[0] += (amplitude_r[ll][n][0]*(float)(ylmr[0]))*rfunc_interp;
                                array_pos[1] += (amplitude_i[ll][n][0]*(float)(ylmr[0]))*rfunc_interp;
//#ifdef DEBUG
//                                if (idx_x==nan_index_x && idx_y==nan_index_y && idx_z==nan_index_z)
//                                {
//                                    printf("ylm[%d][%d] = %.6e .\n", l, 0, ylmr[0] );
//                                    printf("amplitude_r[%d][%d][%d] = %.6e .\n", l, n ,m, amplitude_r[ll][n][0]);
//                                    printf("amplitude_i[%d][%d][%d] = %.6e .\n", l, n ,m, amplitude_i[ll][n][0]);
//                                    printf("rfunc_interp = %.6e .\n", rfunc_interp);
//                                }
//#endif
                            }
                        }
                    }
                } // end of for loop l
//#ifdef DEBUG
//                if (idx_x==(int)(nx/2) && idx_y==(int)(ny/2) && idx_z==(int)(nz/2))
//                    printf("array_r[%d] = %.4e ; array_i[%d] = %.4e \n", index_site, array_pos[0], index_site, array_pos[1]);
//#endif
            } // end of for loop m
            arr_r[index_site] = array_pos[0];
            arr_i[index_site] = array_pos[1];
//#ifdef DEBUG
//            if (idx_x==nan_index_x&&idx_y==nan_index_y&&idx_z==nan_index_z)
//            {
//                printf("arr_r[%d][%d][%d] = %.6e ; arr_i[%d][%d][%d] = %.6e\n", idx_x, idx_y, idx_z, arr_r[index_site], idx_x, idx_y, idx_z, arr_i[index_site]);
//            }
//#endif
        }
    }
    __syncthreads();
#ifdef DEBUG
//    printf("Calculating Expanding ylmr completed on site (%d,%d,%d) .\n", idx_x, idx_y, idx_z);
    if (threadIdx.x==0 && threadIdx.y==0 && threadIdx.z==0 && blockIdx.x==gridDim.x-1 && blockIdx.y==gridDim.y-1)
        printf("Calculating-expanding ylmr completed on grid (%d,%d,%d). \n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif
}

void ylm_init(int l_init,int l_max)
{
    int l,n;
    bool cuda_error_flag;
    float *amplitude_r_n_buff, **amplitude_r_l_buff;
    float *amplitude_i_n_buff, **amplitude_i_l_buff;

    amplitude_host = (cmpx***) malloc((l_max+1-l_init)*sizeof(cmpx**));
    for(l = l_init;l <= l_max;l++)
    {
        amplitude_host[l-l_init] = (cmpx**) malloc((rfunc_num[l-l_init])*sizeof(cmpx*));
        cmpx *amplitude_buf = (cmpx*) malloc((rfunc_num[l-l_init])*(2*l+1)*sizeof(cmpx));
        memset(amplitude_buf, 0, rfunc_num[l-l_init]*(2*l+1)*sizeof(cmpx));

        for(n = 0;n < rfunc_num[l-l_init];n++)
        {
            amplitude_host[l-l_init][n] = amplitude_buf;
            amplitude_buf += (2*l+1);
        }
    }
#ifdef DEBUG
    printf("Memory allocation for amplitude_host completed!\n");
#endif
    
    cuda_error_flag = CUDA_CHECK_ERROR(hipMallocManaged((void**)&amplitude_r, (l_max+1-l_init)*sizeof(float**)));
    if (!cuda_error_flag)
    {
        printf("Allocation for pointer amplitude_r failed!\n");
        exit(1);
    }
#ifdef DEBUG
    else
        printf("Allocation for pointer amplitude_r succeed!\n");
#endif
    cuda_error_flag = CUDA_CHECK_ERROR(hipMallocManaged((void**)&amplitude_i, (l_max+1-l_init)*sizeof(float**)));
    if (!cuda_error_flag)
    {
        printf("Allocation for pointer amplitude_i failed!\n");
        exit(1);
    }
#ifdef DEBUG
    else
        printf("Allocation for pointer amplitude_i succeed!\n");
#endif

    for(l = l_init;l <= l_max;l++)
    {
        cuda_error_flag = CUDA_CHECK_ERROR(hipMallocManaged((void**)&amplitude_r_l_buff, rfunc_num[l-l_init]*sizeof(float*)));
        cuda_error_flag = CUDA_CHECK_ERROR(hipMallocManaged((void**)&amplitude_i_l_buff, rfunc_num[l-l_init]*sizeof(float*)));

//	printf("rank=%d l=%d rfunc_num[%d]=%d\n",rank,l,l-l_init,rfunc_num[l-l_init]);//debug
        for(n = 0;n < rfunc_num[l-l_init];n++)
        {
            cuda_error_flag = CUDA_CHECK_ERROR(hipMallocManaged((void**)&amplitude_r_n_buff, (2*l+1)*sizeof(float)));
            cuda_error_flag = CUDA_CHECK_ERROR(hipMemcpy(&amplitude_r_l_buff[n], &amplitude_r_n_buff, sizeof(float*), hipMemcpyHostToDevice));
//                amplitude[l-l_init][n] = amplitude_buf;
//                amplitude_buf += (2*l+1);
            if (!cuda_error_flag)
            {
                printf("Memory allocation for amplitude_r[%d][%d] failed!\n", l, n);
                exit(1);
            }
#ifdef DEBUG
            else
                printf("Memory allocation for amplitude_r[%d][%d] succeed!\n", l, n);
#endif
            cuda_error_flag = CUDA_CHECK_ERROR(hipMallocManaged((void**)&amplitude_i_n_buff, (2*l+1)*sizeof(float)));
            cuda_error_flag = CUDA_CHECK_ERROR(hipMemcpy(&amplitude_i_l_buff[n], &amplitude_i_n_buff, sizeof(float*), hipMemcpyHostToDevice));
            if (!cuda_error_flag)
            {
                printf("Memory allocation for amplitude_i[%d][%d] failed!\n", l, n);
                exit(1);
            }
#ifdef DEBUG
            else
                printf("Memory allocation for amplitude_i[%d][%d] succeed!\n", l, n);
#endif
        }
        cuda_error_flag = CUDA_CHECK_ERROR(hipMemcpy(&amplitude_r[l-l_init], &amplitude_r_l_buff, sizeof(float**), hipMemcpyHostToDevice));
        if (!cuda_error_flag)
        {
            printf("Memory allocation for amplitude_r[%d] failed!\n", l);
            exit(1);
        }
#ifdef DEBUG
        else
            printf("Memory allocation for amplitude_r[%d] succeed!\n", l);
#endif
        cuda_error_flag = CUDA_CHECK_ERROR(hipMemcpy(&amplitude_i[l-l_init], &amplitude_i_l_buff, sizeof(float**), hipMemcpyHostToDevice));
        if (!cuda_error_flag)
        {
            printf("Memory allocation for amplitude_i[%d] failed!\n", l);
            exit(1);
        }
#ifdef DEBUG
        else
            printf("Memory allocation for amplitude_i[%d] succeed!\n", l);
#endif
    }
    return;
}

void ylm_fin(int l_init,int l_max)
{
    int l;
    bool cuda_error_flag;
    for(l = l_init;l <= l_max;l++)
    {
	int ll=l-l_init;
        free(amplitude_host[ll][0]);
        cuda_error_flag = CUDA_CHECK_ERROR(hipFree(amplitude_r[ll][0]));
        cuda_error_flag = CUDA_CHECK_ERROR(hipFree(amplitude_i[ll][0]));

        free(amplitude_host[ll]);
        cuda_error_flag = CUDA_CHECK_ERROR(hipFree(amplitude_r[ll]));
        cuda_error_flag = CUDA_CHECK_ERROR(hipFree(amplitude_i[ll]));
    }

    free(amplitude_host);
    cuda_error_flag = CUDA_CHECK_ERROR(hipFree(amplitude_r));
    cuda_error_flag = CUDA_CHECK_ERROR(hipFree(amplitude_i));
    printf("Amplitude for spherical harmonics from l_init=%d to l_max=%d free.\n", l_init, l_max);
    return;
}

void store_eigenstates(int l_max, char *filename)
{
    int l,n;
    FILE *pf;
    char str[80];
    int l_init;

    sprintf(str,"%s%d.bin",filename,lnodidx);
    pf = fopen(str, "wb");
    fwrite(&l_init, 1, sizeof(int), pf);
    fwrite(&l_max, 1, sizeof(int), pf);
    fwrite(rfunc_num, l_max+1-l_init, sizeof(int), pf);
    for(l = l_init;l <= l_max;l++)
    {
        fwrite(rv[l-l_init], rfunc_num[l-l_init], sizeof(double), pf);
    	for(n = 0;n < rfunc_num[l-l_init];n++)
    	{
    	    fwrite(rfunc[l-l_init][n], rsiz, sizeof(double), pf);
    	}
    }
    fclose(pf);

    return;
}
void free_egn(int l_max,int l_init)
{
    bool cuda_error_flag;
    
    for(int l=l_init;l<=l_max;l++)
    {
        for (int n=0; n<rfunc_num[l-l_init]; n++) 
        {
            CUDA_CHECK_ERROR(hipFree(rfunc_l_pointer[l-l_init][n]));
        }
        cuda_error_flag = CUDA_CHECK_ERROR(hipFree(rv_pointer[l-l_init]));
        cuda_error_flag = CUDA_CHECK_ERROR(hipFree(rfunc_pointer[l-l_init]));
        free(rfunc_l_pointer[l-l_init]);
    }
    cuda_error_flag = CUDA_CHECK_ERROR(hipFree(rv));
    cuda_error_flag = CUDA_CHECK_ERROR(hipFree(rfunc_num));
    cuda_error_flag = CUDA_CHECK_ERROR(hipFree(rfunc));
    free(rfunc_pointer);
    free(rfunc_l_pointer);
    free(rv_pointer);
    printf("Eigen values and functions from l_init=%d to l_max=%d free.\n", l_init, l_max);//debug
}

void load_eigenstates(int *l_init,int *l_max, int *eigen_num, char *filename,int lnodidx)
{
    int l, n;
    FILE *pf;
    char str[80];
    float *rv_local_f, *rfunc_local_f;
    double *rv_local, *rfunc_local;
    bool cuda_error_flag;
    sprintf(str,"%s%d.bin",filename,lnodidx);
    pf = fopen(str, "rb");
    fread(l_init, 1, sizeof(int), pf);
    fread(l_max, 1, sizeof(int), pf);

    if (*l_init>*l_max)
    {
        printf("l_init > l_max for Lidx=%d ! Pass!\n", Lidx_global);
        return;
    }
    
    *eigen_num = 0;
    rv_pointer = (float**)malloc((*l_max+1-*l_init)*sizeof(float*));
    rfunc_l_pointer = (float***)malloc((*l_max+1-*l_init)*sizeof(float**));
    rfunc_pointer = (float***)malloc((*l_max+1-*l_init)*sizeof(float**));

    cuda_error_flag = CUDA_CHECK_ERROR(hipMallocManaged((void **)&rfunc_num, (*l_max+1-*l_init)*sizeof(int)));//rfunc_num[l] is the number of eigenvalues
    if (!cuda_error_flag)
    {
        printf("Memory allocation for eigen values numbers failed!\n");
        exit(1);
    }
#ifdef DEBUG
    else
        printf("Memory allocation for eigen value number succeed!\n");
#endif
    cuda_error_flag = CUDA_CHECK_ERROR(hipMalloc((void **)&rv, (*l_max+1-*l_init)*sizeof(float*)));//rv[l][n] is the eigen_ln
    if (!cuda_error_flag)
    {
        printf("Memory allocation for eigen values failed!\n");
        exit(1);
    }
#ifdef DEBUG
    else
        printf("Memory allocation for eigen values succeed!\n");
#endif
    cuda_error_flag = CUDA_CHECK_ERROR(hipMalloc((void **)&rfunc, (*l_max+1-*l_init)*sizeof(float**)));//rc[l][n][r] is the eigenfunction_ln
    if (!cuda_error_flag)
    {
        printf("Memory allocation for eigen functions failed!\n");
        exit(1);
    }
#ifdef DEBUG
    else
        printf("Memory allocation for eigen functions succeed!\n");
#endif

    fread(rfunc_num, *l_max+1-*l_init, sizeof(int), pf);

    for(l = *l_init;l <= *l_max;l++)
    {
        int ll = l-*l_init;
        *eigen_num += rfunc_num[ll];

        rv_local = (double*)malloc(rfunc_num[ll]*sizeof(double));
        rv_local_f = (float*)malloc(rfunc_num[ll]*sizeof(float));
        cuda_error_flag = CUDA_CHECK_ERROR(hipMalloc((void **)&rv_pointer[ll], rfunc_num[ll]*sizeof(float)));
        if (!cuda_error_flag)
        {
            printf("Memory allocation for eigen values with l = %d failed!\n", l);
            exit(1);
        }
#ifdef DEBUG
        else
            printf("Memory allocation for eigen values with l = %d succeed!\n", l);
#endif
	fread(rv_local, rfunc_num[ll], sizeof(double), pf);

//        rfunc_pointer[ll] = (float**)malloc(rfunc_num[ll]*sizeof(float*));
        cuda_error_flag = CUDA_CHECK_ERROR(hipMalloc((void**)&rfunc_pointer[ll], rfunc_num[ll]*sizeof(float*)));
        rfunc_l_pointer[ll] = (float**)malloc(rfunc_num[ll]*sizeof(float*));

	for(n = 0;n < rfunc_num[ll];n++)
	{
            rv_local_f[n] = (float)(rv_local[n]);
            rfunc_local = (double*)malloc(rsiz*sizeof(double));
            rfunc_local_f = (float*)malloc(rsiz*sizeof(float));

            cuda_error_flag = CUDA_CHECK_ERROR(hipMalloc((void **)&rfunc_l_pointer[ll][n], rsiz*sizeof(float)));

	    fread(rfunc_local, rsiz, sizeof(double), pf);
            for (int r=0; r<rsiz; r++)
                rfunc_local_f[r] = (float)(rfunc_local[r]);

            cuda_error_flag = CUDA_CHECK_ERROR(hipMemcpy(rfunc_l_pointer[ll][n], rfunc_local_f, rsiz*sizeof(float), hipMemcpyHostToDevice));
            if (!cuda_error_flag)
            {
                printf("Memory copy for eigen functions to rfunc_pointer with l = %d and n = %d failed!\n", l, n);
                exit(1);
            }
#ifdef DEBUG
            else
                printf("Memory copy for eigen functions with l = %d and n = %d succeed!\n", l, n);
#endif
            free(rfunc_local_f);
            free(rfunc_local);
	}

        cuda_error_flag = CUDA_CHECK_ERROR(hipMemcpy(rfunc_pointer[ll], rfunc_l_pointer[ll], rfunc_num[ll]*sizeof(float*), hipMemcpyHostToDevice));
        if (!cuda_error_flag)
        {
            printf("Memory copy for eigen functions to rfunc_pointer with l = %d failed!\n", l);
            exit(1);
        }
#ifdef DEBUG
        else
            printf("Memory copy for eigen functions to rfunc_pointer with l = %d succeed!\n", l);
#endif

        cuda_error_flag = CUDA_CHECK_ERROR(hipMemcpy(rv_pointer[ll], rv_local_f, rfunc_num[ll]*sizeof(float), hipMemcpyHostToDevice));
        if (!cuda_error_flag)
        {
            printf("Memory copy for eigen values to rv_pointer with l = %d failed!\n", l);
            exit(1);
        }
#ifdef DEBUG
        else
            printf("Memory copy for eigen values to rv_pointer with l = %d succeed!\n", l);
#endif
        
        free(rv_local_f);
        free(rv_local);
//        *ln_total_count += rfunc_num[ll];
    }
    cuda_error_flag = CUDA_CHECK_ERROR(hipMemcpy(rv, rv_pointer, (*l_max+1-*l_init)*sizeof(float*), hipMemcpyHostToDevice));
    if (!cuda_error_flag)
    {
        printf("Memory copy for eigen values failed!\n");
        exit(1);
    }
#ifdef DEBUG
    else
        printf("Memory copy for eigen functions succeed!\n", l);
#endif
    cuda_error_flag = CUDA_CHECK_ERROR(hipMemcpy(rfunc, rfunc_pointer, (*l_max+1-*l_init)*sizeof(float**), hipMemcpyHostToDevice));
    if (!cuda_error_flag)
    {
        printf("Memory copy for eigen functions failed!\n");
        exit(1);
    }
#ifdef DEBUG
    else
        printf("Memory copy for eigen functions succeed!\n", l);
#endif
    fclose(pf);
    return;
}

void store_ylm(int l_init, int l_max, char *filename)
{
    int l;
    FILE *pf;
    char filen[80];

    sprintf(filen,"%s%s%d%s",filename,"L",lnodidx,".bin");
    pf = fopen(filen, "wb");

    for(l = l_init;l <= l_max;l++)
    {
	int ll=l-l_init;
        void *addr_of_buf = amplitude_host[ll][0];
        int   size_of_buf = rfunc_num[ll] * (2 * l + 1) * 2;
        fwrite(addr_of_buf, sizeof(double), size_of_buf, pf);
    }
    fclose(pf);

    return;
}

void load_ylm(int l_init, int l_max, char *filename, int lnodidx)
{
    int l, n, m;
    FILE *pf;
    char filen[80];

    sprintf(filen,"%s%s%d%s",filename,"L",lnodidx,".bin");
    pf = fopen(filen, "rb");
    
    for(l = l_init;l <= l_max;l++)
    {
        void *addr_of_buf = amplitude_host[l-l_init][0];
        int   size_of_buf = rfunc_num[l-l_init] * (2 * l + 1) * 2;
        fread(addr_of_buf, sizeof(double), size_of_buf, pf);
#ifdef DEBUG
        printf("Amplitude of L=%d have been loaded to amplitude_host .\n", l);
#endif
        for (n=0; n<rfunc_num[l-l_init]; n++)
        {
            for (m=0; m<2*l+1; m++)
            {
//                printf("%.4e\t%.4e\n", amplitude_host[l-l_init][n][m][0], amplitude_host[l-l_init][n][m][1]);
                amplitude_r[l-l_init][n][m] = (float)(amplitude_host[l-l_init][n][m][0]);
#ifdef DEBUG
                printf("Amplitude of L=%d, n=%d, m=%d have been copied to amplitude_r.\n", l, n, m);
#endif
                amplitude_i[l-l_init][n][m] = (float)(amplitude_host[l-l_init][n][m][1]);
#ifdef DEBUG
                printf("Amplitude of L=%d, n=%d, m=%d have been copied to amplitude_i.\n", l, n, m);
#endif
            }
#ifdef DEBUG
            printf("Amplitude of L=%d, n= %d have been copied to amplitude_r and amplitude_i.\n", l, n);
#endif
        }
#ifdef DEBUG
        printf("Amplitude of L=%d have been copied to amplitude_r and amplitude_i.\n", l);
#endif
    }
    fclose(pf);
//

//#ifdef DEBUG
//    int test_l = 110 , test_n = 10, test_m = 88;
//    int ll = test_l-l_init;
//    printf("Amplitude Test:\n");
//    if (test_m!=0)
//    {
////        printf("Amp[%d][%d][%d] = %.4e%+.4e ; Amp[%d][%d][%d] = %.4e%+.4e .\n", test_l, test_n, test_m, amplitude_host[ll][test_n][test_m][0], amplitude_hostl[ll][test_n][test_m][1], test_l, test_n, 2*(test_l)+1-test_m, amplitude_host[ll][test_n][2*test_l+1-test_m][0], amplitude_host[ll][test_n][2*test_l+1-test_m][1]);
//        printf("Amp[%d][%d][%d] = %.4e%+.4e ; Amp[%d][%d][%d] = %.4e%+.4e .\n", test_l, test_n, test_m, amplitude_r[ll][test_n][test_m], amplitude_i[ll][test_n][test_m], test_l, test_n, 2*(test_l)+1-test_m, amplitude_r[ll][test_n][2*test_l+1-test_m], amplitude_i[ll][test_n][2*test_l+1-test_m]);
//    }
//    else
//    {
////        printf("Amp[%d][%d][%d] = %.4e%+.4e .\n", test_l, test_n, test_m, amplitude_host[ll][test_n][test_m][0], amplitude_host[ll][test_n][test_m][1]);
//        printf("Amp[%d][%d][%d] = %.4e%+.4e .\n", test_l, test_n, test_m, amplitude_r[ll][test_n][test_m], amplitude_i[ll][test_n][test_m]);
//    }
//#endif
    return;
}

//void show_ylm(int l_init,int l_max, char *filename,int iter)
//{
//    FILE *pf;
//    FILE *pf_text;
//    int n,l,m;
//    char filen[80];
//    
//    int lnodidx=rank%lnod;
//
//    if(rank == lnodidx)
//    {
//	sprintf(filen,"%s%s%d%s",filename,"L",lnodidx,".bin");
//        pf = fopen(filen, "wb");
//    }
//    if(rank == lnodidx)
//    {
//// change the way naming the file 2020.12.28
////        sprintf(filen,"%s%d%s","amp_rconL",lnodidx,".txt");
//        sprintf(filen,"%s%s%d%s",filename,"L",lnodidx,".txt");
////
////		printf("filename=%s\n",filen);//debug
//        pf_text = fopen(filen, "w");
//    }
//
//    for(l = l_init;l <= l_max;l++)
//        for(n = 0;n < rfunc_num[l-l_init];n++)
//            for(m = 0; m < l * 2 + 1;m++)
//            {
//                int mm;
//                mm = m > l ? m - (2*l+1) : m;
//                double eigenvalue = rv[l-l_init][n];
//                double amp_r = amplitude [l-l_init][n][m][0];
//                double amp_i = amplitude [l-l_init][n][m][1];
//                double amp_r_out = amp_r;
//                double amp_i_out = amp_i;
////                MPI_Reduce(&amp_r, &amp_r_out, 1, MPI_DOUBLE, MPI_SUM, 0, MPI_COMM_WORLD);
////                MPI_Reduce(&amp_i, &amp_i_out, 1, MPI_DOUBLE, MPI_SUM, 0, MPI_COMM_WORLD);
////                MPI_Barrier(MPI_COMM_WORLD);
//                if(rank == lnodidx)
//                {
//                    fwrite(&l ,         sizeof(int),    1, pf);
//                    fwrite(&n ,         sizeof(int),    1, pf);
//                    fwrite(&mm,         sizeof(int),    1, pf);
//                    fwrite(&eigenvalue, sizeof(double), 1, pf);
//                    fwrite(&amp_r_out , sizeof(double), 1, pf);
//                    fwrite(&amp_i_out , sizeof(double), 1, pf);
//
//                    fprintf(pf_text, "%5d ", l );
//                    fprintf(pf_text, "%5d ", n );
////  change the order of n and mm 2020.12.28
//                    fprintf(pf_text, "%5d ", mm);
////
//                    fprintf(pf_text, "%15.10le ", eigenvalue);
//                    fprintf(pf_text, "%15.10le ", amp_r_out );
//                    fprintf(pf_text, "%15.10le ", amp_i_out );
//                    fprintf(pf_text, "\n");
//                }
//            }
//    if(rank == lnodidx)
//        fclose(pf);
//    if(rank == lnodidx)
//        fclose(pf_text);
//
//    MPI_Barrier(MPI_COMM_WORLD);
//
//    return;
//}
//void do_expand_ylm(float *arr_r, float *arr_i, float *calculate_timer, float *expand_timer, int l_max,int l_init, int ln_total_count, bool timing_flag)
void do_expand_ylm(float *arr_r, float *arr_i, float *cal_and_expand_timer, int l_max,int l_init, bool timing_flag)
{
//    int size_shared = tpB*tpB*tpB*sizeof(float)*ln_total_count;
//    if (size_shard>64000)
//    {
//        printf("Size needed by shared memory too large! Exit!\n");
//        exit(1);
//    }
    hipEvent_t start, stop;
    bool cuda_error_flag;
    float time_temp;
    dim3 tpB(tpB_x, tpB_y, tpB_z);
    dim3 bpG(bpG_x, bpG_y, bpG_z);

    if (timing_flag)
    {
        cuda_error_flag = CUDA_CHECK_ERROR(hipEventCreate(&start));
        cuda_error_flag = CUDA_CHECK_ERROR(hipEventCreate(&stop));
        cuda_error_flag = CUDA_CHECK_ERROR(hipEventRecord(start,0));
    }

#ifdef OCTANT_DECOMPOSE
    cal_and_expand_ylm_O<<<bpG,tpB>>>(rsiz, nx, ny, nz, octant, (float)dx, (float)dy, (float)dz, (float)xcen, (float)ycen, (float)zcen, (float)dr, (float)r_max, arr_r, arr_i, rfunc_num, rfunc, amplitude_r, amplitude_i, l_max, l_init);
#else
    cal_and_expand_ylm<<<bpG,tpB>>>(rsiz, nx, ny, nz, (float)dx, (float)dy, (float)dz, (float)xcen, (float)ycen, (float)zcen, (float)dr, (float)r_max, arr_r, arr_i, rfunc_num, rfunc, amplitude_r, amplitude_i, l_max, l_init);
#endif

    cuda_error_flag = CUDA_CHECK_ERROR(hipPeekAtLastError());
    if (!cuda_error_flag)
    {
        int Lidx_temp = Lidx_global-1;
        printf("Dump with Lidx = %d .\n", Lidx_temp);
        write_array(true,&Lidx_temp,file_r,file_i);
        strcpy(file_r, buff_r);
        strcpy(file_i, buff_i);
//        exit(1);
    }
#ifdef DEBUG
    else
        printf("Calculating-expanding ylm succeed!\n");
#endif
    cuda_error_flag = CUDA_CHECK_ERROR(hipDeviceSynchronize());

    if (timing_flag)
    {
        cuda_error_flag = CUDA_CHECK_ERROR(hipEventRecord(stop,0));
        cuda_error_flag = CUDA_CHECK_ERROR(hipEventSynchronize(stop));
        cuda_error_flag = CUDA_CHECK_ERROR(hipEventElapsedTime(&time_temp, start, stop));
        printf("Time for expanding ylm is %.4e s.\n", time_temp/1000.);
        *cal_and_expand_timer += time_temp/1000.;
    }
//	printf("end of do_expand_ylm,rank=%d\n",rank);//debug
    return;
}


/*
 The primary function in this library.
 This function generates all the Ylm(\theta,\phi) values up to l = l_max, for a given x,y,z coordinate.
 The x,y,z vector is normalised to unit length first.
 Values are calculated using iterative method provided by new version NR,
 which would prevent unstatability underflow issues.
 */

///*
// * generate amplitudes from King model
// */
//void gen_amp(int l_init,int l_max,double A,double beta,double Ec,double mu0,double amp_gs_r,double amp_gs_i,bool israndom)
//{
//    	int lnodidx=rank%lnod;
//	double Glo_totalmass=0;
//	if(rank==0) printf("mu0=%e\n",mu0);
//	
//	for(int masscount=0;masscount<2;masscount++){
//		double totalmass=0;
////		if(rank==0) printf("A=%e,beta=%e,Ec=%e,eta0=%e\n",A,beta,Ec,eta0);//debug
//		for(int l=l_init;l<=l_max;l++){
//			int lpr=l-l_init;
//			void *addr_of_buf = amplitude[lpr][0];
//			int   size_of_buf = rfunc_num[l-l_init] * (2 * l + 1) * 2;
//			if(rank==lnodidx)
//			{
//			    for(int n=0;n<rfunc_num[l-l_init];n++){
//				for(int m=0;m<l*2+1;m++){
////					double std=King(rv[l][n],A,beta,Ec);
//					double std=Fermionic_King(rv[l-l_init][n],A,beta,Ec,mu0);
////					std=sqrtf(std/2.);//2(std^2)=DF(E)
////					printf("std=%e\n",std);//debug
//					if(l==0&&n==0){//ground state
//						amplitude[lpr][n][m][0]=amp_gs_r;//real part of amplitude
//						amplitude[lpr][n][m][1]=amp_gs_i;//imaginary part
//					}
//					else if(israndom){
//	//				continue;//debug
//						amplitude[lpr][n][m][0]=rand_normal(0.0,sqrtf(std/2.));//real part of amplitude
//						amplitude[lpr][n][m][1]=rand_normal(0.0,sqrtf(std/2.));//imaginary part
//					}
//					else{
//						amplitude[lpr][n][m][0]=sqrtf(std/2.);//real part of amplitude
//						amplitude[lpr][n][m][1]=sqrtf(std/2.);//imaginary part
////						printf("hi\n");//debug
//					}
//				        totalmass+=(amplitude[lpr][n][m][0]*amplitude[lpr][n][m][0]+
//	   		                amplitude[lpr][n][m][1]*amplitude[lpr][n][m][1]);
////					printf("totalmass=%e\n",totalmass);//debug
//				}
//			    }
//			}
////			MPI_Bcast(addr_of_buf, size_of_buf, MPI_DOUBLE, lnodidx, MPI_COMM_WORLD);
////			printf("l=%d rank=%d\n",l,rank);//debug
//		}
//        	MPI_Barrier(MPI_COMM_WORLD);
//   		MPI_Reduce(&totalmass,&Glo_totalmass, 1, MPI_DOUBLE,MPI_SUM, 0, MPI_COMM_WORLD);
//		
//		if(rank==0)
//		{
//  		        printf("global total mass=%e\n",Glo_totalmass);//print total mass
////			printf("global total mass=%e; gs_mass=%e\n",Glo_totalmass, amplitude[0][0][0][0]*amplitude[0][0][0][0]+amplitude[0][0][0][1]*amplitude[0][0][0][1]);//print total mass and core_mass up to r_vir, for debug
//	   	        A=A*halomass/Glo_totalmass;
//		}
//        	MPI_Barrier(MPI_COMM_WORLD);
//		MPI_Bcast(&A, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
//	}
//        MPI_Barrier(MPI_COMM_WORLD);
//	return;
//}
void read_amp(int l_max,char *filename)
{
    FILE *pf;
    int n,l,m;

    pf = fopen(filename, "rb");

    for(l = 0;l <= l_max;l++)
        for(n = 0;n < rfunc_num[l];n++)
            for(m = 0; m < l * 2 + 1;m++)
            {
//		printf("l=%d n=%d m=%d\n",l,n,m);//debug
                int mm,ll,nn;
                mm = m > l ? m - (2*l+1) : m;
                double eigenvalue;
                double amp_r_in;
                double amp_i_in;

              	fread(&ll,         sizeof(int),    1, pf);
              	fread(&nn,         sizeof(int),    1, pf);
              	fread(&mm,         sizeof(int),    1, pf);
              	fread(&eigenvalue, sizeof(double), 1, pf);
              	fread(&amp_r_in  , sizeof(double), 1, pf);
             	fread(&amp_i_in  , sizeof(double), 1, pf);

                amplitude_host [l][n][m][0] = (float)(amp_r_in);
                amplitude_host [l][n][m][1] = (float)(amp_i_in);
            }
    fclose(pf);
    return;
};//end read_amp
