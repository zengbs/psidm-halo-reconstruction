#include "hip/hip_runtime.h"
/* move extern int Lidx_global and extern int octant_global to arr.h  2020.12.29*/
/* fix typo in cal_dis_O() 2020.12.30 */
/* add timing in array_add_ylm() 2020.12.31 */
/* add r_max debug in array_add_ylm() 2020.12.31 */
/* add auto renaming output file in write_array() 2021.01.03 */
/* add iter_test for testing the code serval times 2021.01.07*/
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include"macros.h"
#include"text_io.h"
#include"main.h"
#include"arr.h"
#include"ylm.h"
#define RESTART_FILENAME_R "RESTART_R"
#define RESTART_FILENAME_I "RESTART_I"

void run_single_lidx(void);

void run(void)
{
    run_single_lidx();
}

void run_single_lidx(void)
{
    float time_temp;
    char buff_r[128], buff_i[128];
    
//    solve_init();
//    if(rank==0)
//        gen_r(r_1,r);
//    MPI_Barrier(MPI_COMM_WORLD);
//    MPI_Bcast(r, rsiz, MPI_DOUBLE, 0, MPI_COMM_WORLD);
//    MPI_Bcast(r_1, rsiz, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    //printf("r[0]=%e 0\n",r[0]);//debug
	
    int l_max=0;
    int l_init=0;
    int counter=1;
    
/// call r_max ///
#ifdef OCTANT_DECOMPOSE
    cal_r_max_O();
#else
    cal_r_max();
#endif
#ifdef DEBUG
    printf("r_max found! r_max = %.4e .\n", r_max);
#endif
    if (dump_flag==1)
    {
        strcpy(buff_r, file_r);
        strcpy(buff_i, file_i);
    }

    array_init();
    if (restart_flag==1)
    {
        read_array(true, restart_id, RESTART_FILENAME_R, RESTART_FILENAME_I);
        printf("Restart from Lidx=%d .\n", restart_id);
    }
    else
        restart_id = 0;

#ifdef DEBUG
    printf("CUDA halo reconstruction starts...\n");
#endif

    hipEvent_t start, stop, eigen_copy_start, eigen_copy_stop, amplitude_copy_start, amplitude_copy_stop, lidx_start, lidx_stop;
    hipSetDevice(0);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&eigen_copy_start);
    hipEventCreate(&eigen_copy_stop);
    hipEventCreate(&amplitude_copy_start);
    hipEventCreate(&amplitude_copy_stop);
    hipEventCreate(&lidx_start);
    hipEventCreate(&lidx_stop);
    hipEventRecord(start,0);
    	
// add data dump and restart 2021.01.04
    for(int lidx=restart_id;lidx<lnod;lidx++)
    {
        Lidx_global = lidx;
        hipEventRecord(lidx_start,0);
        hipEventRecord(eigen_copy_start,0);
        
        /*** load eigenstates and broadcast to every nodes***/
        load_eigenstates(&l_init,&l_max,"egn",Lidx_global);
#ifdef DEBUG
        printf("Eigen states loading completed!\n");
#endif
        hipEventRecord(eigen_copy_stop,0);
        hipEventSynchronize(eigen_copy_stop);
        hipEventElapsedTime(&time_temp, eigen_copy_start, eigen_copy_stop);
        printf("Total time for copying eigen states from l_init=%d to l_max=%d is %.4e s.\n", l_init, l_max, time_temp/1000.);
//  ignoring all operations if l_max<l_init 2020.12.29
        if (l_max<l_init)
        {
            printf("l_init = %d @ Lidx = %d will be ignored since l_max < l_init !\n", l_init, Lidx_global);
            continue;
        }
#ifdef DEBUG
        printf("Start kernel preparation...\n");
#endif
        /*** generate upper bound ***/
//        gen_ubound(rv[0][0] * 0., l_max,l_init);
        ylm_init(l_init,l_max);
#ifdef DEBUG
        printf("Memory allocation for spherical harmonics completed.\n");
#endif

        hipEventRecord(amplitude_copy_start,0);
        /*** read amplitudes and Bcast ***/
        load_ylm(l_init,l_max,"amp_only",Lidx_global);
        /*** generate density in simulation box ***/
#ifdef DEBUG
        printf("Amplitudes of spherical harmonics loading completed.\n");
#endif
        hipEventRecord(amplitude_copy_stop,0);
        hipEventSynchronize(amplitude_copy_stop);
        hipEventElapsedTime(&time_temp, amplitude_copy_start, amplitude_copy_stop);
        printf("Total time for copying  amplitudes from l_init=%d to l_max=%d is %.4e s.\n", l_init, l_max, time_temp/1000.);

        array_add_ylm(l_max,l_init,true);

        hipDeviceSynchronize();
        ylm_fin(l_init,l_max);
        free_egn(l_max,l_init);

        if (dump_flag==1&&(counter%dump_interval)==0)
        {
            printf("===============================================\n");
            printf("Data dumped at Lidx=%d.\n", Lidx_global);
            printf("===============================================\n");
            write_array(true,&Lidx_global,file_r,file_i);
            strcpy(file_r, buff_r);
            strcpy(file_i, buff_i);
        }

        hipEventRecord(lidx_stop,0);
        hipEventSynchronize(lidx_stop);
        hipEventElapsedTime(&time_temp, lidx_start, lidx_stop);
        printf("Total time for running through l_init=%d to l_max=%d takes %.4e s.\n",l_init, l_max, time_temp/1000.);

        counter ++;
//        hipDeviceReset();
    }
    printf("Output 3D wave function...\n");
    write_array(false,&Lidx_global,file_r,file_i);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_temp, start, stop);
    time_temp /= 1000.;
    int total_hr = (int)(time_temp/3600);
    int total_min = (int)((time_temp-3600*total_hr)/60);
    double total_sec = time_temp - 3600.*total_hr - 60.*total_min;
    printf("Total computation time for reconstructing halo is %d hr %d min %.2f s .\n", total_hr, total_min, total_sec);

    array_fin();
    return;
}
