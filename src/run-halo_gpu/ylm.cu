#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<string.h>
#include<hip/hip_runtime.h>
#include"macros.h"
#include"main.h"
#include"ylm.h"
#include"arr.h"

int lnodidx = Lidx_global;

__device__ void cal_index_r(int rsiz, float r, float dr, int *index_r, float *frac_r)
{
    float rr = r/dr;
    int i = (int)floor(rr);
    if(i >= rsiz)
        i = rsiz - 1;
    if(i <= 0)
        i = 1;
    *frac_r = rr - i;
    *index_r = i;
    return;
}

#ifdef OCTANT_DECOMPOSE
__global__ void cal_and_expand_ylm_O(int rsiz, int nx, int ny, int nz, int octant, float dx, float dy, float dz, float xcen, float ycen, float zcen, float dr, float r_max, float *arr_r, float *arr_i, int *rfunc_num, double ***rfunc, cmpx ***amplitude, int l_max,int l_init)
#else
__global__ void cal_and_expand_ylm(int rsiz, int nx, int ny, int nz, float dx, float dy, float dz, float xcen, float ycen, float zcen, float dr, float r_max, float *arr_r, float* arr_i, int *rfunc_num, double *** rfunc, cmpx ***amplitude, int l_max, int l_init)
#endif
{
//if (blockIdx.x==0&&blockIdx.y==0&&blockIdx.z==0)
//    printf("%d, %s\n", __LINE__, __FUNCTION__);
    int idx_x = threadIdx.x + blockIdx.x*blockDim.x;
    int idx_y = threadIdx.y + blockIdx.y*blockDim.y;
    int idx_z = threadIdx.z + blockIdx.z*blockDim.z;

//    extern __shared__ float rfunc_interp[];

    if (idx_x<nx&&idx_y<ny&&idx_z<nz)
    {
        float x, y, z, r;
#ifdef OCTANT_DECOMPOSE
        cal_dis_x_O(idx_x, nx, octant, dx, xcen, &x);
        cal_dis_y_O(idx_y, ny, octant, dy, ycen, &y);
        cal_dis_z_O(idx_z, nz, octant, dz, zcen, &z);
#else
        cal_dis_x(idx_x, dx, xcen, &x);
        cal_dis_y(idx_y, dy, ycen, &y);
        cal_dis_z(idx_z, dz, zcen, &z);
#endif
        cal_dis_r(&x, &y, &z, &r);        
//#ifdef DEBUG
//        if (idx_x==(int)(nx/2)&&idx_y==(int)(ny/2)&&idx_z==(int)(nz/2))
//        {
//            printf("Kernel executing...\n");
//            printf("Calculating distance r completed.\n");
//            printf("x = %.4e , y = %.4e , z= %.4e , r = %.4e , r_max = %.4e .\n", x, y, z, r, r_max);
//        }
//#endif
        if (r<r_max)
        {
            int n, l, m, ll;
            int index_r, index_site;
            float frac_r, rfunc_interp;
            float fact_1, fact_2 = 0., oldfact = 0., temp;  // for ylm computation
            float pmm_new, pmm_old, pml, pml_lm1, phr_new, phr_old, phi_new, phi_old;
            float ylmr[2], array_pos[2];

            cal_index_r(rsiz, r, dr, &index_r, &frac_r);
            index_site = get_pos(idx_x, idx_y, idx_z, nx, ny);
//            index_thread = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
//#ifdef DEBUG
//            if (idx_x==(int)(nx/2)&&idx_y==(int)(ny/2)&&idx_z==(int)(nz/2))
//                printf("index_site = %d ; index_r = %d ; frac_r = %.4e .\n", index_site, index_r, frac_r);
//#endif
            array_pos[0] = (float)0.;
            array_pos[1] = (float)0.;

            x /= r;
            y /= r;
            z /= r;
            fact_1 = -(float)1.;

//            int counter = 0;
//            int shift = ln_total_count*index_thread;
//            for (l=0; l<=l_max-l_init;l++)
//            {
//                for (n=0; n<rfunc[l]; n++)
//                    *(rfunc_interp+shift+counter+n) = (float)(rfunc[ll][n][index_r-1])*(1-frac_r) + (float)(rfunc[ll][n][index_r])*(frac_r);
//                counter += rfunc[l];
//            }

            for (m=0; m<=l_max; m++)
            {
                if (m==0)
                {
                    phr_new = (float)1.;
                    phi_new = (float)0.;
                    pmm_new = (float)1.;
                    pmm_old = (float)1.;
                }
                else
                {
                    if (m==1)
                    {
                        phr_new = x;
                        phi_new = y;
                        phr_old = phr_new;
                        phi_old = phi_new;
                    }
                    else
                    {
                        phr_new = phr_old*x - phi_old*y;
                        phi_new = phi_old*x + phr_old*y;
                        phr_old = phr_new;
                        phi_old = phi_new;
                    }
                    pmm_new = fact_1 * pmm_old / (fact_1+(float)1.);
                    pmm_old = pmm_new;
                }
                fact_1 += (float)2.;
        
                pmm_new = sqrtf(((float)2.*m+(float)1.)*pmm_new/((float)4.*(float)M_PI));
        
                for(l = m;l <= l_max; l++)
                {
                    if(l > m)
                        fact_2 = sqrtf(((float)4.*l*l-(float)1.)/(l*l-m*m));
                    if(l == m)
                        pml = pmm_new;
                    else if(l == m+(float)1.)
                    {
                        pml_lm1 = pml;
                        pml = fact_2*z*pml;
//#ifdef DEBUG
//                        if (idx_x==0 && idx_y==0 && idx_z==0)
//                        {
//                            if (m==0&&l==1)
//                                printf("fact_2 = %.6e ; z = %.6e ; pml_lm1 = %.6e ; pml = %.6e .\n", fact_2, z, pml_lm1, fact_2*z*pml_lm1);
//                        }
//#endif
                    }
                    else
                    {
                        temp = pml;
                        pml = fact_2*(z*pml-pml_lm1/oldfact);
                        pml_lm1 = temp;
                    }
                    oldfact = fact_2;
                    if(m!=0)
                    {
                        ylmr[0] = pml * phr_new * sqrtf((float)2.); // ylmr[0] = ylm[l][m]
                        ylmr[1] = pml * phi_new * sqrtf((float)2.); // ylmr[1] = ylmr[l][-m+2*l+1]
                        ll = l-l_init;
                        if (ll>=0)
                        {
                            for (n=0; n<rfunc_num[ll]; n++)
                            {
                                rfunc_interp = (float)(rfunc[ll][n][index_r-1])*((float)1.-frac_r) + (float)(rfunc[ll][n][index_r])*(frac_r);
                                array_pos[0] += ((float)(amplitude[ll][n][m][0])*ylmr[0]+(float)(amplitude[ll][n][-m+2*l+1][0])*ylmr[1])*rfunc_interp;
                                array_pos[1] += ((float)(amplitude[ll][n][m][1])*ylmr[0]+(float)(amplitude[ll][n][-m+2*l+1][1])*ylmr[1])*rfunc_interp;
//#ifdef DEBUG
//                                if (idx_x==(int)(nx/2)&&idx_y==(int)(ny/2)&&idx_z==(int)(nz/2))
//                                {
//                                    if (m==6&&ll==2&&n==10)
//                                        printf("amplitude[%d][%d][%d] = %.6e%+.6e ; amplitude[%d][%d][%d] = %.6e%+.6e ; ylmr[%d][%d] = %.6e ; ylmr[%d][%d] = %.6e ; rfunc_interp = %.6e .\n", l, n, m, (float)(amplitude[ll][n][m][0]), (float)(amplitude[ll][n][m][1]), l, n, -m+2*l+1, (float)(amplitude[ll][n][-m+2*l+1][0]), (float)(amplitude[ll][n][m][1]), l, m, ylmr[0], l, -m+2*l+1, ylmr[1], rfunc_interp);
//                                }
//#endif
                            }
                        }
                    }
                    else
                    {
                        ylmr[0] = pml * phr_new;
                        ll = l-l_init;
                        if (ll>=0)
                        {
                            for (n=0; n<rfunc_num[ll]; n++)
                            {
                                rfunc_interp = (float)(rfunc[ll][n][index_r-1])*(1-frac_r) + (float)(rfunc[ll][n][index_r])*(frac_r);
                                array_pos[0] += ((float)(amplitude[ll][n][0][0])*ylmr[0])*rfunc_interp;
                                array_pos[1] += ((float)(amplitude[ll][n][0][1])*ylmr[0])*rfunc_interp;
                            }
                        }
                    }
//#ifdef DEBUG
//                    if (idx_x==0 && idx_y==0 && idx_z==0)
//                    {
//                        if (m!=0)
//                        {
//                            printf("ylm[%d][%d] = %.6e .\n", l, m, ylmr[0] );
//                            printf("ylm[%d][%d] = %.6e .\n", l, -m+2*l+1, ylmr[1] );
//                        }
//                        else
//                            printf("ylm[%d][%d] = %.6e .\n", l, 0, ylmr[0] );
//                    }
//#endif
                } // end of for loop l
//#ifdef DEBUG
//                if (idx_x==(int)(nx/2) && idx_y==(int)(ny/2) && idx_z==(int)(nz/2))
//                    printf("array_r[%d] = %.4e ; array_i[%d] = %.4e \n", index_site, array_pos[0], index_site, array_pos[1]);
//#endif
            } // end of for loop m
            arr_r[index_site] = array_pos[0];
            arr_i[index_site] = array_pos[1];
        }
    }
    __syncthreads();
#ifdef DEBUG
//    printf("Calculating Expanding ylmr completed on site (%d,%d,%d) .\n", idx_x, idx_y, idx_z);
    if (threadIdx.x==0 && threadIdx.y==0 && threadIdx.z==0 && blockIdx.x==gridDim.x-1 && blockIdx.y==gridDim.y-1)
        printf("Calculating-expanding ylmr completed on grid (%d,%d,%d). \n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif
}

void ylm_init(int l_init,int l_max)
{
    hipError_t err;
    err = hipMallocManaged((void**)&amplitude, (l_max+1-l_init)*sizeof(cmpx**));
//    hipMallocManaged((void**)&amplitude, (l_max+1-l_init)*sizeof(cmpx**));
    if (err!=hipSuccess)
    {
        printf("Allocation for pointer amplitufe failed!\n");
        exit(1);
    }
#ifdef DEBUG
    else
        printf("Allocation for pointer amplitude succeed!\n");
#endif

    int l,n;
    float *amplitude_n_buff, **amplitude_l_buff;
    for(l = l_init;l <= l_max;l++)
    {
        err = hipMallocManaged((void**)&amplitude_l_buff, rfunc_num[l-l_init]*sizeof(cmpx*));
//        cmpx *amplitude_buf = (cmpx*) malloc((rfunc_num[l-l_init])*(2*l+1)*sizeof(cmpx));
//        memset(amplitude_buf, 0, rfunc_num[l-l_init]*(2*l+1)*sizeof(cmpx));

//	printf("rank=%d l=%d rfunc_num[%d]=%d\n",rank,l,l-l_init,rfunc_num[l-l_init]);//debug
        for(n = 0;n < rfunc_num[l-l_init];n++)
        {
            err = hipMallocManaged((void**)&amplitude_n_buff, (2*l+1)*sizeof(cmpx));
            err = hipMemcpy(&amplitude_l_buff[n], &amplitude_n_buff, sizeof(cmpx*), hipMemcpyHostToDevice);
//                amplitude[l-l_init][n] = amplitude_buf;
//                amplitude_buf += (2*l+1);
            if (err!=hipSuccess)
            {
                printf("Memory allocation for amplitude[%d][%d] failed!\n", l, n);
                exit(1);
            }
#ifdef DEBUG
            else
                printf("Memory allocation for amplitude[%d][%d] succeed!\n", l, n);
#endif
        }
        err = hipMemcpy(&amplitude[l-l_init], &amplitude_l_buff, sizeof(cmpx**), hipMemcpyHostToDevice);
        if (err!=hipSuccess)
        {
            printf("Memory allocation for amplitude[%d] failed!\n", l);
            exit(1);
        }
#ifdef DEBUG
        else
            printf("Memory allocation for amplitude[%d] succeed!\n", l);
#endif
    }
    return;
}

void ylm_fin(int l_init,int l_max)
{
    int l;
    for(l = 0;l <= l_max;l++)
    {
	int ll=l-l_init;
//        for (int n=0; n<rfunc_num[ll]; n++)
//        {
//            hipFree(amplitude[ll][n]);
//            free(amplitude_host[ll][n]);
//        }

        hipFree(amplitude[ll]);
//        free(amplitude_host[ll]);
    }
    hipFree(amplitude);
//    free(amplitude_host);
    printf("Amplitude for spherical harmonics from l_init=%d to l_max=%d free.\n", l_init, l_max);
    return;
}

void store_eigenstates(int l_max, char *filename)
{
    int l,n;
    FILE *pf;
    char str[80];
    int l_init;

    sprintf(str,"%s%d.bin",filename,lnodidx);
    pf = fopen(str, "wb");
    fwrite(&l_init, 1, sizeof(int), pf);
    fwrite(&l_max, 1, sizeof(int), pf);
    fwrite(rfunc_num, l_max+1-l_init, sizeof(int), pf);
    for(l = l_init;l <= l_max;l++)
    {
        fwrite(rv[l-l_init], rfunc_num[l-l_init], sizeof(double), pf);
    	for(n = 0;n < rfunc_num[l-l_init];n++)
    	{
    	    fwrite(rfunc[l-l_init][n], rsiz, sizeof(double), pf);
    	}
    }
    fclose(pf);

    return;
}
void free_egn(int l_max,int l_init)
{
    for(int l=l_init;l<=l_max;l++)
    {
//			printf("rank=%d rfunc_num[%d]=%d\n",rank,l-l_init,rfunc_num[l-l_init]);//debug
        for(int n=0;n<rfunc_num[l-l_init];n++)
            hipFree(rfunc[l-l_init][n]);
        hipFree(rv[l-l_init]);
        hipFree(rfunc[l-l_init]);
    }
    hipFree(rv);
    hipFree(rfunc_num);
    hipFree(rfunc);
    printf("Eigen values and functions from l_init=%d to l_max=%d free.\n", l_init, l_max);//debug
}

//void load_eigenstates(int *l_init,int *l_max, int *ln_total_count, char *filename,int lnodidx)
void load_eigenstates(int *l_init,int *l_max, char *filename,int lnodidx)
{
    int l,n;
    FILE *pf;
    char str[80];
    double *rv_buff, *rfunc_n_buff, **rfunc_l_buff;
    hipError_t err;
    sprintf(str,"%s%d.bin",filename,lnodidx);
    pf = fopen(str, "rb");
    fread(l_init, 1, sizeof(int), pf);
    fread(l_max, 1, sizeof(int), pf);
//    *ln_total_count = 0;

    err = hipMallocManaged((void **)&rfunc_num, (*l_max+1-*l_init)*sizeof(int));//rfunc_num[l] is the number of eigenvalues
    if (err!=hipSuccess)
    {
        printf("Memory allocation for eigen value number failed!\n");
        exit(1);
    }
#ifdef DEBUG
    else
        printf("Memory allocation for eigen value number succeed!\n");
#endif
    hipMallocManaged((void **)&rv, (*l_max+1-*l_init)*sizeof(double*));//rv[l][n] is the eigen_ln
    hipMallocManaged((void **)&rfunc, (*l_max+1-*l_init)*sizeof(double**));//rc[l][n][r] is the eigenfunction_ln

    fread(rfunc_num, *l_max+1-*l_init, sizeof(int), pf);
    for(l = *l_init;l <= *l_max;l++)
    {
        int ll = l-*l_init;
        hipMallocManaged((void **)&rv_buff, (*l_max+1-*l_init)*sizeof(double));
        err = hipMemcpy(&rv[ll], &rv_buff, sizeof(double*), hipMemcpyHostToDevice);
        if (err!=hipSuccess)
        {
            printf("Memory allocation for eigen values with l = %d failed!\n", l);
            exit(1);
        }
#ifdef DEBUG
        else
            printf("Memory allocation for eigen values with l = %d succeed!\n", l);
#endif
	fread(rv[ll], rfunc_num[ll], sizeof(double), pf);

        hipMallocManaged((void **)&rfunc_l_buff, rfunc_num[ll]*sizeof(double*));
        hipMemcpy(&rfunc[ll], &rfunc_l_buff, sizeof(double**), hipMemcpyHostToDevice);

	for(n = 0;n < rfunc_num[ll];n++)
	{
            hipMallocManaged((void **)&rfunc_n_buff, rsiz*sizeof(double));
            err = hipMemcpy(&rfunc[ll][n], &rfunc_n_buff, sizeof(double*), hipMemcpyHostToDevice);
            if (err!=hipSuccess)
            {
                printf("Memory allocation for eigen functions with l = %d , n = %d failed!\n", l, n);
                exit(1);
            }
//#ifdef DEBUG
//            else
//                printf("Memory allocation for eigen functions with l = %d , n = %d succeed!\n", l, n);
//#endif
	    fread(rfunc[ll][n], rsiz, sizeof(double), pf);
	}
        if (err!=hipSuccess)
        {
            printf("Memory allocation for eigen functions with l = %d failed!\n", l);
            exit(1);
        }
#ifdef DEBUG
        else
            printf("Memory allocation for eigen functions with l = %d succeed!\n", l);
//        *ln_total_count += rfunc_num[ll];
#endif
    }
    fclose(pf);
    return;
}

void store_ylm(int l_init, int l_max, char *filename)
{
    int l;
    FILE *pf;
    char filen[80];

    sprintf(filen,"%s%s%d%s",filename,"L",lnodidx,".bin");
    pf = fopen(filen, "wb");

    for(l = l_init;l <= l_max;l++)
    {
	int ll=l-l_init;
        void *addr_of_buf = amplitude[ll][0];
        int   size_of_buf = rfunc_num[ll] * (2 * l + 1) * 2;
        fwrite(addr_of_buf, sizeof(double), size_of_buf, pf);
    }
    fclose(pf);

    return;
}

void load_ylm(int l_init, int l_max, char *filename,int lnodidx)
{
    int l, n;
    FILE *pf;
    char filen[80];
    cmpx ***amplitude_local = (cmpx***)malloc((l_max+1-l_init)*sizeof(cmpx**));
    
//  initailize amplitude_local
    for(l=0; l<=l_max-l_init;l++)
    {
        amplitude_local[l] = (cmpx**) malloc((rfunc_num[l])*sizeof(cmpx*));
        cmpx *amplitude_buf = (cmpx*) malloc((rfunc_num[l])*(2*(l+l_init)+1)*sizeof(cmpx));

//      printf("rank=%d l=%d rfunc_num[%d]=%d\n",rank,l,l-l_init,rfunc_num[l-l_init]);//debug
        for(n = 0;n < rfunc_num[l];n++)
        {
            amplitude_local[l][n] = amplitude_buf;
            amplitude_buf += (2*(l+l_init)+1);
        }
    }
//

// read from bin file and copy to unified memory
    sprintf(filen,"%s%s%d%s",filename,"L",lnodidx,".bin");
    pf = fopen(filen, "rb");
    for(l = l_init;l <= l_max;l++)
    {
        void *addr_of_buf = amplitude_local[l-l_init][0];
        int   size_of_buf = rfunc_num[l-l_init] * (2 * l + 1) * 2;
        fread(addr_of_buf, sizeof(double), size_of_buf, pf);
        for (n=0; n<rfunc_num[l-l_init]; n++)
        {
            for (int m=0; m<2*l+1; m++)
            {
                amplitude[l-l_init][n][m][0] = amplitude_local[l-l_init][n][m][0];
                amplitude[l-l_init][n][m][1] = amplitude_local[l-l_init][n][m][1];
            }
        }
        free(amplitude_local[l-l_init]);
    }
    free(amplitude_local);
    fclose(pf);
//

//#ifdef DEBUG
//    int test_l = 110 , test_n = 10, test_m = 88;
//    int ll = test_l-l_init;
//    printf("Amplitude Test:\n");
//    if (test_m!=0)
//    {
////        printf("Amp[%d][%d][%d] = %.4e%+.4e ; Amp[%d][%d][%d] = %.4e%+.4e .\n", test_l, test_n, test_m, amplitude_local[ll][test_n][test_m][0], amplitude_local[ll][test_n][test_m][1], test_l, test_n, 2*(test_l)+1-test_m, amplitude_local[ll][test_n][2*test_l+1-test_m][0], amplitude_local[ll][test_n][2*test_l+1-test_m][1]);
//        printf("Amp[%d][%d][%d] = %.4e%+.4e ; Amp[%d][%d][%d] = %.4e%+.4e .\n", test_l, test_n, test_m, amplitude[ll][test_n][test_m][0], amplitude[ll][test_n][test_m][1], test_l, test_n, 2*(test_l)+1-test_m, amplitude[ll][test_n][2*test_l+1-test_m][0], amplitude[ll][test_n][2*test_l+1-test_m][1]);
//    }
//    else
//    {
////        printf("Amp[%d][%d][%d] = %.4e%+.4e .\n", test_l, test_n, test_m, amplitude_local[ll][test_n][test_m][0], amplitude_local[ll][test_n][test_m][1]);
//        printf("Amp[%d][%d][%d] = %.4e%+.4e .\n", test_l, test_n, test_m, amplitude[ll][test_n][test_m][0], amplitude[ll][test_n][test_m][1]);
//    }
//#endif
    return;
}

//void show_ylm(int l_init,int l_max, char *filename,int iter)
//{
//    FILE *pf;
//    FILE *pf_text;
//    int n,l,m;
//    char filen[80];
//    
//    int lnodidx=rank%lnod;
//
//    if(rank == lnodidx)
//    {
//	sprintf(filen,"%s%s%d%s",filename,"L",lnodidx,".bin");
//        pf = fopen(filen, "wb");
//    }
//    if(rank == lnodidx)
//    {
//// change the way naming the file 2020.12.28
////        sprintf(filen,"%s%d%s","amp_rconL",lnodidx,".txt");
//        sprintf(filen,"%s%s%d%s",filename,"L",lnodidx,".txt");
////
////		printf("filename=%s\n",filen);//debug
//        pf_text = fopen(filen, "w");
//    }
//
//    for(l = l_init;l <= l_max;l++)
//        for(n = 0;n < rfunc_num[l-l_init];n++)
//            for(m = 0; m < l * 2 + 1;m++)
//            {
//                int mm;
//                mm = m > l ? m - (2*l+1) : m;
//                double eigenvalue = rv[l-l_init][n];
//                double amp_r = amplitude [l-l_init][n][m][0];
//                double amp_i = amplitude [l-l_init][n][m][1];
//                double amp_r_out = amp_r;
//                double amp_i_out = amp_i;
////                MPI_Reduce(&amp_r, &amp_r_out, 1, MPI_DOUBLE, MPI_SUM, 0, MPI_COMM_WORLD);
////                MPI_Reduce(&amp_i, &amp_i_out, 1, MPI_DOUBLE, MPI_SUM, 0, MPI_COMM_WORLD);
////                MPI_Barrier(MPI_COMM_WORLD);
//                if(rank == lnodidx)
//                {
//                    fwrite(&l ,         sizeof(int),    1, pf);
//                    fwrite(&n ,         sizeof(int),    1, pf);
//                    fwrite(&mm,         sizeof(int),    1, pf);
//                    fwrite(&eigenvalue, sizeof(double), 1, pf);
//                    fwrite(&amp_r_out , sizeof(double), 1, pf);
//                    fwrite(&amp_i_out , sizeof(double), 1, pf);
//
//                    fprintf(pf_text, "%5d ", l );
//                    fprintf(pf_text, "%5d ", n );
////  change the order of n and mm 2020.12.28
//                    fprintf(pf_text, "%5d ", mm);
////
//                    fprintf(pf_text, "%15.10le ", eigenvalue);
//                    fprintf(pf_text, "%15.10le ", amp_r_out );
//                    fprintf(pf_text, "%15.10le ", amp_i_out );
//                    fprintf(pf_text, "\n");
//                }
//            }
//    if(rank == lnodidx)
//        fclose(pf);
//    if(rank == lnodidx)
//        fclose(pf_text);
//
//    MPI_Barrier(MPI_COMM_WORLD);
//
//    return;
//}
//void do_expand_ylm(float *arr_r, float *arr_i, float *calculate_timer, float *expand_timer, int l_max,int l_init, int ln_total_count, bool timing_flag)
void do_expand_ylm(float *arr_r, float *arr_i, float *cal_and_expand_timer, int l_max,int l_init, bool timing_flag)
{
//    int size_shared = tpB*tpB*tpB*sizeof(float)*ln_total_count;
//    if (size_shard>64000)
//    {
//        printf("Size needed by shared memory too large! Exit!\n");
//        exit(1);
//    }
    hipEvent_t start, stop;
    hipError_t err;
    float time_temp;
    dim3 tpB(tpB_x, tpB_y, tpB_z);
    dim3 bpG(bpG_x, bpG_y, bpG_z);

    if (timing_flag)
    {
        err = hipEventCreate(&start);
        err = hipEventCreate(&stop);
        err = hipEventRecord(start,0);
    }
    printf("Peek error befor kernel launchded: %s .\n", hipGetErrorString(err));  

#ifdef OCTANT_DECOMPOSE
    cal_and_expand_ylm_O<<<bpG,tpB>>>(rsiz, nx, ny, nz, octant, (float)dx, (float)dy, (float)dz, (float)xcen, (float)ycen, (float)zcen, (float)dr, (float)r_max, arr_r, arr_i, rfunc_num, rfunc, amplitude, l_max, l_init);
#else
    cal_and_expand_ylm<<<bpG,tpB>>>(rsiz, nx, ny, nz, (float)dx, (float)dy, (float)dz, (float)xcen, (float)ycen, (float)zcen, (float)dr, (float)r_max, arr_r, arr_i, rfunc_num, rfunc, amplitude, l_max, l_init);
#endif

    err = hipPeekAtLastError();
    if (err!=hipSuccess)
    {
        int Lidx_temp = Lidx_global-1;
        printf("Calculating-exapnding ylm failed due to error: %s !\n", hipGetErrorString(err));
        printf("Dump with Lidx = %d .\n", Lidx_temp);
        write_array(true,&Lidx_temp,file_r,file_i);
        exit(1);
    }
#ifdef DEBUG
    else
        printf("Calculating-expanding ylm succeed!\n");
#endif
    hipDeviceSynchronize();

    if (timing_flag)
    {
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time_temp, start, stop);
        printf("Time for expanding ylm is %.4e s.\n", time_temp/1000.);
        *cal_and_expand_timer += time_temp/1000.;
    }
//	printf("end of do_expand_ylm,rank=%d\n",rank);//debug
    return;
}


/*
 The primary function in this library.
 This function generates all the Ylm(\theta,\phi) values up to l = l_max, for a given x,y,z coordinate.
 The x,y,z vector is normalised to unit length first.
 Values are calculated using iterative method provided by new version NR,
 which would prevent unstatability underflow issues.
 */

///*
// * generate amplitudes from King model
// */
//void gen_amp(int l_init,int l_max,double A,double beta,double Ec,double mu0,double amp_gs_r,double amp_gs_i,bool israndom)
//{
//    	int lnodidx=rank%lnod;
//	double Glo_totalmass=0;
//	if(rank==0) printf("mu0=%e\n",mu0);
//	
//	for(int masscount=0;masscount<2;masscount++){
//		double totalmass=0;
////		if(rank==0) printf("A=%e,beta=%e,Ec=%e,eta0=%e\n",A,beta,Ec,eta0);//debug
//		for(int l=l_init;l<=l_max;l++){
//			int lpr=l-l_init;
//			void *addr_of_buf = amplitude[lpr][0];
//			int   size_of_buf = rfunc_num[l-l_init] * (2 * l + 1) * 2;
//			if(rank==lnodidx)
//			{
//			    for(int n=0;n<rfunc_num[l-l_init];n++){
//				for(int m=0;m<l*2+1;m++){
////					double std=King(rv[l][n],A,beta,Ec);
//					double std=Fermionic_King(rv[l-l_init][n],A,beta,Ec,mu0);
////					std=sqrtf(std/2.);//2(std^2)=DF(E)
////					printf("std=%e\n",std);//debug
//					if(l==0&&n==0){//ground state
//						amplitude[lpr][n][m][0]=amp_gs_r;//real part of amplitude
//						amplitude[lpr][n][m][1]=amp_gs_i;//imaginary part
//					}
//					else if(israndom){
//	//				continue;//debug
//						amplitude[lpr][n][m][0]=rand_normal(0.0,sqrtf(std/2.));//real part of amplitude
//						amplitude[lpr][n][m][1]=rand_normal(0.0,sqrtf(std/2.));//imaginary part
//					}
//					else{
//						amplitude[lpr][n][m][0]=sqrtf(std/2.);//real part of amplitude
//						amplitude[lpr][n][m][1]=sqrtf(std/2.);//imaginary part
////						printf("hi\n");//debug
//					}
//				        totalmass+=(amplitude[lpr][n][m][0]*amplitude[lpr][n][m][0]+
//	   		                amplitude[lpr][n][m][1]*amplitude[lpr][n][m][1]);
////					printf("totalmass=%e\n",totalmass);//debug
//				}
//			    }
//			}
////			MPI_Bcast(addr_of_buf, size_of_buf, MPI_DOUBLE, lnodidx, MPI_COMM_WORLD);
////			printf("l=%d rank=%d\n",l,rank);//debug
//		}
//        	MPI_Barrier(MPI_COMM_WORLD);
//   		MPI_Reduce(&totalmass,&Glo_totalmass, 1, MPI_DOUBLE,MPI_SUM, 0, MPI_COMM_WORLD);
//		
//		if(rank==0)
//		{
//  		        printf("global total mass=%e\n",Glo_totalmass);//print total mass
////			printf("global total mass=%e; gs_mass=%e\n",Glo_totalmass, amplitude[0][0][0][0]*amplitude[0][0][0][0]+amplitude[0][0][0][1]*amplitude[0][0][0][1]);//print total mass and core_mass up to r_vir, for debug
//	   	        A=A*halomass/Glo_totalmass;
//		}
//        	MPI_Barrier(MPI_COMM_WORLD);
//		MPI_Bcast(&A, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
//	}
//        MPI_Barrier(MPI_COMM_WORLD);
//	return;
//}
void read_amp(int l_max,char *filename)
{
    FILE *pf;
    int n,l,m;

    pf = fopen(filename, "rb");

    for(l = 0;l <= l_max;l++)
        for(n = 0;n < rfunc_num[l];n++)
            for(m = 0; m < l * 2 + 1;m++)
            {
//		printf("l=%d n=%d m=%d\n",l,n,m);//debug
                int mm,ll,nn;
                mm = m > l ? m - (2*l+1) : m;
                double eigenvalue;
                double amp_r_in;
                double amp_i_in;

              	fread(&ll,         sizeof(int),    1, pf);
              	fread(&nn,         sizeof(int),    1, pf);
              	fread(&mm,         sizeof(int),    1, pf);
              	fread(&eigenvalue, sizeof(double), 1, pf);
              	fread(&amp_r_in  , sizeof(double), 1, pf);
             	fread(&amp_i_in  , sizeof(double), 1, pf);

                amplitude [l][n][m][0] = (float)(amp_r_in);
                amplitude [l][n][m][1] = (float)(amp_i_in);
            }
    fclose(pf);
    return;
};//end read_amp
