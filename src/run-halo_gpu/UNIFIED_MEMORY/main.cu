#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<unistd.h>
#include<time.h>
#include<math.h>
#include<stdbool.h>
#include<hip/hip_runtime.h> 
#include"main.h"
#include"arr.h"
#include"ylm.h"
#include"macros.h"
#include"text_io.h"
#include"solve_eigenvalues.h"
#include"extension.h"
//

using namespace std; 

/*
 global parameters
 */

int Lidx_global=-1;
int octant_global=-1;
int eigen_num;

double mass      = 0.;
double newton_g  = 0.;
double planck_h  = 0.;
double eta       = 0.;
double red_shift = 0.;
double scale_fac = 0.;

extern double newton_g_mks;
extern double mass_of_sun;
extern double kpc_to_meter;
extern double electron_charge;
extern double speed_of_light;
extern double hbar_mks;

// adding data dumping and restarting 2021.01.04
int dump_flag = -1;
int dump_interval = 0;
int dump_id = 0;
int restart_flag = -1;
int restart_id = 0;
// adding cosmology parameters 2020.12.10
double Omega_m0  = 0.;
double Omega_lambda0  = 0.;
double h0 = 0.;
//
// adding newly defined parameters 2020.12.10
double m_a_22 = 0.;
double mass_core_ratio = 0.;
double phase_between_RI = 0.;  // in unit of 2\pi
double potential_scale = 0.;
//
// adding NFW parameters 2020.12.10
double NFW_C = 0.;
double NFW_A = 0.;
double NFW_rho_0 = 0.;
double r_vir = 0.;
double r_e = 0.;
double R_s = 0.;
//
// adding convergence criteria 2020.12.14
double criteria = 0.;
//
// adding contrast factor for determining virial radius 2020.12.16
double CF = 0.;
//
// adding GPU parameters 2021.01.09
int tpB_x, tpB_y, tpB_z;
int bpG_x, bpG_y, bpG_z;
int N_site;
//

double box_range = 0.;
double r_range   = 0.;
double den_vir   = 0.;
double r_eigen   =0.;//energy cut= V(r_eigen)

double box_dr = 0.;
double dx = box_dr;
double dy = box_dr;
double dz = box_dr;

double box_dr_cnt = 0.;
double dx_cnt;
double dy_cnt;
double dz_cnt;

int scal_cnt = 1;
double xoff_cnt = 0.;
double yoff_cnt = 0.;
double zoff_cnt = 0.;


double dr;
double r_max;

/*
box size
*/
int nbox = 0;
int nbox2= 0;
int nx = nbox;
int ny = nbox;
int nz = nbox;

int nbox_cnt = 0;
int nx_cnt;
int ny_cnt;
int nz_cnt;
/*
the total grid points of the eigenstates solver
maximum l number
maximum states number per l mode
*/
int rsiz = 4096;
int lsiz = 100;
int Lsiz = 100;
int nsiz = 100;
int lnod = 1;
//modify lsiz calculation (2020.12.25)
int l_rest = 0; 

double xcen;
double ycen;
double zcen;
/*
 array file
*/
char   file_r[strsiz];
char   file_i[strsiz];

char   file_r_cnt[strsiz];
char   file_i_cnt[strsiz];
/* initial potential guess file */
char   init_pot[strsiz];
/* final potential output file */
char   final_pot[strsiz];
/* convergence error output file */
//char   convergence_file[strsiz];

/*
 global variables
*/

//cmpxf  *array     = NULL;
//cmpxf  *array_cnt = NULL;
int    *rfunc_num = NULL;
float **rv       = NULL;
float ***rfunc   = NULL;
//double **rv       = NULL;
//double ***rfunc   = NULL;
float *array_r_host, *array_i_host;
float ***amplitude_r, ***amplitude_i;

cmpx ***amplitude_host;
//int *rfunc_num_host;
//double **rv_host;
//double ***rfunc_host;

int ubound_l   = -1;
int *ubound_n  = NULL;
int lbound_l   = -1;
int *lbound_n  = NULL;
/*
char filename_amp[strsiz] = "amp";
char filename_ylm[strsiz] = "ylm";
char filename_eigen[strsiz] = "egn";
*/
/*
Fermionic King model parameters.
distribution function:	f(E)=A(exp(E-Ec)-1)/(1+A/mu0*exp(E-Ec))
 */
double A=0.;
double beta=0.;
double Ec=0.;
double mu0=0.;
double amp_gs_r=0.;  // will be M_soliton_r(r_e)*cos(phase)
double amp_gs_i=0.;  // will be M_soliton_r(r_e)*sin(phase)

double halomass=0.;
double perturb_frac=0.;
double r_c=0.;

int iter_i=0;
int iter_max=12;
int octant=0;

int main(int argc, char** argv)
{
//	long seed = 123456789;
	long seed = 49651525;

#       ifdef OCTANT_DECOMPOSE
        printf( "Octant decomposition: ON\n" );
#       else
        printf( "Octant decomposition: OFF\n" );
#       endif

//	clock_t t;
	srand(seed);
//	srand(time(NULL));

//	read_para("parameter_r.txt");//read parameters

  	read_para( argv[1] );

        octant_global = octant;

//        if ( lnod > size )
//        {
//           fprintf( stderr, "ERROR : LNOD (%d) > # of MPI ranks (%d) !!\n", lnod, size );
//           MPI_Abort( MPI_COMM_WORLD, 1 );
//        }

	run();
	
	return 0;
}

void set_para(char *paraname, char *paravalue, char *readname, char *readvalue, char type)
{
        if(strcmp(readname, paraname) == 0)
        {
            if(type == 'i')
                *((int    *)paravalue) = atoi(readvalue);
            if(type == 'd')
                *((double *)paravalue) = atof(readvalue);
            if(type == 's')
                strcpy (paravalue, readvalue);
        }
}

//void gen_arti_halo(clock_t t,int l_init,int l_max)
//{
//	/// generate amplitudes ///
//	gen_amp(l_init,l_max,A,beta,Ec,mu0,amp_gs_r,amp_gs_i,false);//gen amp in rank 0 and Bcast to all nodes
//	printf("amplitude finished\n");
//	
//	/// call r_max ///
//	cal_r_max();
//
//	/// generate upper bound ///
//	gen_ubound(rv[0][0] * 0., l_max,l_init);
//
//	/// construct density profile ///
//	t=clock();
//	den_profile(rho,l_init,l_max);
//	t=clock()-t;
//	printf("density profile takes %e minutes, rank=%d\n",((float)t)/CLOCKS_PER_SEC/60.,rank);
//
//	/// generate artificial halo density profile and potential ///
//	//bool isartihalo=true;
//	//gen_output(rho, isartihalo);
//
//	return;
//}//end of gen_arti_halo

void read_para(const char *parafile)
{

    FILE *pf;
        pf = fopen(parafile,"r");

    while(1)
    {
        char paraname [strsiz];
        char paravalue[strsiz];
        int ret;
        ret = fscanf(pf,"%s%s", paraname, paravalue);
        if(ret == EOF)
            break;
        if(ret != 2)
            continue;

        set_para("MASS"      , (char *)(& mass      ), paraname, paravalue, 'd');
//        set_para("NEWTON_G"  , (char *)(& newton_g  ), paraname, paravalue, 'd');
        set_para("PLANCK_H"  , (char *)(& planck_h  ), paraname, paravalue, 'd');
//        set_para("ETA"       , (char *)(& eta       ), paraname, paravalue, 'd');
        set_para("RED_SHIFT" , (char *)(& red_shift ), paraname, paravalue, 'd');
//	set_para("SCALE_FAC" , (char *)(& scale_fac ), paraname, paravalue, 'd');
	set_para("FILE_R"    , (char *)(& file_r    ), paraname, paravalue, 's');
	set_para("FILE_I"    , (char *)(& file_i    ), paraname, paravalue, 's');
//      set_para("FILE_R_CNT", (char *)(& file_r_cnt), paraname, paravalue, 's');
//	set_para("FILE_I_CNT", (char *)(& file_i_cnt), paraname, paravalue, 's');
        set_para("BOX_RANGE" , (char *)(& box_range ), paraname, paravalue, 'd');
        set_para("R_RANGE"   , (char *)(& r_range   ), paraname, paravalue, 'd');
        set_para("R_EIGEN"   , (char *)(& r_eigen   ), paraname, paravalue, 'd');
//	set_para("DEN_VIR"   , (char *)(& den_vir   ), paraname, paravalue, 'd');
        set_para("NBOX"      , (char *)(& nbox      ), paraname, paravalue, 'i');
        set_para("NBOX2"     , (char *)(& nbox2     ), paraname, paravalue, 'i');
        set_para("RSIZ"      , (char *)(& rsiz      ), paraname, paravalue, 'i');
        set_para("LSIZ"      , (char *)(& Lsiz      ), paraname, paravalue, 'i');
        set_para("NSIZ"      , (char *)(& nsiz      ), paraname, paravalue, 'i');
        set_para("LNOD"      , (char *)(& lnod      ), paraname, paravalue, 'i');
// A will be calculated automatically 2020.12.21
//        set_para("A"         , (char *)(& A         ), paraname, paravalue, 'd');
        set_para("BETA"      , (char *)(& beta      ), paraname, paravalue, 'd');
        set_para("ESCAPE_E"  , (char *)(& Ec        ), paraname, paravalue, 'd');
        set_para("MU0"       , (char *)(& mu0      ), paraname, paravalue, 'd');
        set_para("HALOMASS"  , (char *)(& halomass  ), paraname, paravalue, 'd');
//        set_para("XCEN"      , (char *)(& xcen      ), paraname, paravalue, 'd');
//        set_para("YCEN"      , (char *)(& ycen      ), paraname, paravalue, 'd');
//        set_para("ZCEN"      , (char *)(& zcen      ), paraname, paravalue, 'd');
//        set_para("GSAMP_R"   , (char *)(& amp_gs_r  ), paraname, paravalue, 'd');
//        set_para("GSAMP_I"   , (char *)(& amp_gs_i  ), paraname, paravalue, 'd');
        set_para("PERTURB"   , (char *)(&perturb_frac),paraname, paravalue, 'd');
//        set_para("CORE_R"    , (char *)(&r_c        ), paraname, paravalue, 'd');
        set_para("ITER_I"    , (char *)(&iter_i     ), paraname, paravalue, 'i');
        set_para("ITER_MAX"  , (char *)(&iter_max   ), paraname, paravalue, 'i');
        set_para("OCTANT"    , (char *)(&octant     ), paraname, paravalue, 'i');
//	set_para("NBOX_CNT"  , (char *)(& nbox_cnt  ), paraname, paravalue, 'i');
//	set_para("SCAL_CNT"  , (char *)(& scal_cnt  ), paraname, paravalue, 'i');
	set_para("INIT_POT"  , (char *)(& init_pot  ), paraname, paravalue, 's');
// adding final potential output file 2020.12.27
	set_para("FINAL_POT"  , (char *)(& final_pot  ), paraname, paravalue, 's');
// adding parameters 2020.12.10
	set_para("OMEGA_M0"  , (char *)(& Omega_m0  ), paraname, paravalue, 'd');
//	set_para("OMEGA_LAMBDA0"  , (char *)(& Omega_lambda0  ), paraname, paravalue, 'd');
	set_para("SMALL_H0"  , (char *)(& h0  ), paraname, paravalue, 'd');
	set_para("MASS_CORE_RATIO"  , (char *)(& mass_core_ratio  ), paraname, paravalue, 'd');
	set_para("PHASE_BETWEEN_RI"  , (char *)(& phase_between_RI  ), paraname, paravalue, 'd');
	set_para("NFW_A"  , (char *)(& NFW_A  ), paraname, paravalue, 'd');
	set_para("NFW_C"  , (char *)(& NFW_C  ), paraname, paravalue, 'd');
// adding convergence check 2020.12.14
	set_para("CRITERIA"  , (char *)(& criteria  ), paraname, paravalue, 'd');
//	set_para("CONVERGENCE_FILE"  , (char *)(& convergence_file  ), paraname, paravalue, 's');
//
// adding contrast factor for determing virial radius 2020.12.16
	set_para("CONTRAST_FACTOR"  , (char *)(& CF  ), paraname, paravalue, 'd');
//
// adding data dumping and restarting 2021.01.04
	set_para("DUMP_FLAG"  , (char *)(& dump_flag  ), paraname, paravalue, 'i');
	set_para("DUMP_INTERVAL"  , (char *)(& dump_interval  ), paraname, paravalue, 'i');
	set_para("RESTART_FLAG"  , (char *)(& restart_flag  ), paraname, paravalue, 'i');
	set_para("RESTART_ID"  , (char *)(& restart_id  ), paraname, paravalue, 'i');
//       
// adding GPU parameters 2021.01.09
	set_para("THREADS_PER_BLOCK_X"  , (char *)(& tpB_x  ), paraname, paravalue, 'i');
	set_para("THREADS_PER_BLOCK_Y"  , (char *)(& tpB_y  ), paraname, paravalue, 'i');
	set_para("THREADS_PER_BLOCK_Z"  , (char *)(& tpB_z  ), paraname, paravalue, 'i');
//	set_para("BLOCKS_PER_GRID"  , (char *)(& bpG  ), paraname, paravalue, 'i');
//        
    }
    fclose(pf);

    nx = nbox;
    ny = nbox;
    nz = nbox;
    nx_cnt = nbox_cnt;
    ny_cnt = nbox_cnt;
    nz_cnt = nbox_cnt;

    box_dr = box_range / nbox;
    dx = box_dr;
    dy = box_dr;
    dz = box_dr;
    box_dr_cnt = box_dr / scal_cnt;
    dx_cnt = box_dr_cnt;
    dy_cnt = box_dr_cnt;
    dz_cnt = box_dr_cnt;
    xoff_cnt = (nx - 1) * dx / 2 - (nx_cnt - 1) * dx_cnt / 2;
    yoff_cnt = (ny - 1) * dy / 2 - (ny_cnt - 1) * dy_cnt / 2;
    zoff_cnt = (nz - 1) * dz / 2 - (nz_cnt - 1) * dz_cnt / 2;

    bpG_x = (nbox+tpB_x-1)/tpB_x;
    bpG_y = (nbox+tpB_y-1)/tpB_y;
    bpG_z = (nbox+tpB_z-1)/tpB_z;
    N_site = nx*ny*nz;
    dr = r_range / rsiz;
    scale_fac = 1./(1.+red_shift);
    phase_between_RI *= 2.*M_PI;
//    determing xcen, ycen, zcen automatically 2020.12.29
    srand(3413567);
    float randnum;
    randnum = 5e-3*(((float)rand()/RAND_MAX)-0.5);
    xcen = box_range*(0.5+randnum);
    randnum = 5e-3*(((float)rand()/RAND_MAX)-0.5);
    ycen = box_range*(0.5+randnum);
    randnum = 5e-3*(((float)rand()/RAND_MAX)-0.5);
    zcen = box_range*(0.5+randnum);
//
//    modify lsiz calculation 2020.12.25
    l_rest = (Lsiz+1)%lnod;
//    debug
//    printf("%.4f\t%.4f\t%.4f\n", halomass, mass_core_ratio, red_shift);
//    adding calculation of m_a_22 2020.12.10
    m_a_22 = mass/1e-22;
//    adding calculation of Omega_lambda0 2020.12.10
    Omega_lambda0 = 1.-Omega_m0;
//    adding calculation of r_c 2020.12.10
    r_c = r_core();  // in unit of Mpc/h
//    printf("Core radius is %.16e Mpc/h.\n", r_c);
//    adding calculation of NFW parameters
    NFW_A = log(1+NFW_C)-NFW_C/(1.+NFW_C);
    double H0 = h0*100./kpc_to_meter;                                                                           //  km/s/Mpc converted to unit of s^-1
    double rho_critical = 3.*H0*H0/8./M_PI/newton_g_mks;                                                        // in unit of kg/m^3
    double rho_critical_z_matter = (rho_critical*Omega_m0)/pow(1./(1.+red_shift),3.);                           // in unit of kg/m^3
    r_vir = 1e-3*h0*pow((halomass*1e11*mass_of_sun/(4./3.*M_PI*CF*rho_critical_z_matter)),1./3.)/kpc_to_meter; // in unit of Mpc/h
    NFW_rho_0 = halomass*1e11/(4.*M_PI*pow(r_vir/NFW_C/r_c,3.)*NFW_A);                                          // in unit of 1e11 M_sun/r_c^3
    R_s = r_vir/NFW_C/r_c;                                                                                      // in unit of r_c
    r_e = find_re_bisection(rho_soliton_r, rho_NFW_r, 1.*r_c, 10.*r_c);                                         // in unit of Mpc/h
    double M_soliton_at_re = M_soliton_r(r_e);
    amp_gs_r = sqrt(M_soliton_at_re)*cos(phase_between_RI);
    amp_gs_i = sqrt(M_soliton_at_re)*sin(phase_between_RI);
    newton_g = (3.*Omega_m0*pow(H0,2.)/(8.*M_PI))/pow(H0,2.);
    potential_scale = newton_g*scale_fac/(r_c)/newton_g_mks;
    eta = pow(newton_g*pow(hbar_mks/(m_a_22*1e-22*electron_charge/pow(speed_of_light,2.))/scale_fac,2.)/newton_g_mks/(pow(kpc_to_meter*1e3/h0,4.)*(rho_critical*Omega_m0)),-0.5);

// automatically calculate A 2020.12.21
    A = pow(2.*pow(dr/h0*1e3*kpc_to_meter,3.)*rho_critical*Omega_m0/mass_of_sun/1e11,0.5);
//
// determine r_eigen by virial radius if R_EIGEN is not specified 2021.01.04 */
    if (r_eigen==0.)
        r_eigen = r_vir*1.01;
//

    printf("Input Parameters:\n");
    printf("R_RANGE = %.8e\n", r_range);
    printf("BOX_RANGE = %.8e\n", box_range);
    printf("R_EIGEN = %.8e\n", r_eigen);
    printf("NBOX = %d\n", nbox);
    printf("RSIZ = %d\n", rsiz);
    printf("LSIZ = %d\n", Lsiz);
    printf("NSIZ = %d\n", nsiz);
    printf("LNOD = %d\n", lnod);
    printf("A = %.8e\n", A);
    printf("BETA = %.8e\n", beta);
    printf("ESCAPE_E = %.8e\n", Ec);
    printf("MU0 = %.8e\n", mu0);
    printf("HALO_MASS = %.8e\n", halomass);
    printf("PERTURB = %.8e\n", perturb_frac);
    printf("OMEGA_M0 = %.8e\n", Omega_m0);
    printf("SMALL_H0 = %.8e\n", h0);
    printf("MASS_CORE_RATIO = %.8e\n", mass_core_ratio);
    printf("NFW_C = %.8e\n", NFW_C);
    printf("PHASE_BETWEEN_RI = %.8e\n", phase_between_RI);
    printf("CRITERIA = %.8e\n", criteria);
    printf("CONTRAST_FACTOR = %.8e\n", CF);
# ifdef OCTANT_DECOMPOSE
    printf("OCTANT = %d\n", octant);
# endif
    printf("DUMP_FLAG = %d\n", dump_flag);
    printf("DUMP_INTERVAL = %d\n", dump_interval);
    printf("RESTART_FLAG = %d\n", restart_flag);
    printf("RESTART_ID = %d\n", restart_id);
    printf("THREADS_PER_BLOCK_X = %d\n", tpB_x);
    printf("THREADS_PER_BLOCK_Y = %d\n", tpB_y);
    printf("THREADS_PER_BLOCK_Z = %d\n", tpB_z);
    printf("BLOCKS_PER_GRID_X = %d\n", bpG_x);
    printf("BLOCKS_PER_GRID_Y = %d\n", bpG_y);
    printf("BLOCKS_PER_GRID_Z = %d\n", bpG_z);
    printf("=================================================\n");
    printf("Calculated Parameters:\n");
    printf("ETA = %.8e\n", eta);
    printf("CORE_R = %.8e\n", r_c);
    printf("VIRIAL_R = %.8e ; POTENTIAL_VIRIAL = %.8e .\n", r_vir, phi_total_r(r_vir));
    printf("EQUAL_R = %.8e\n", r_e);
    printf("GSAMPR = %.8e ; GSAMP_I = %.8e ; squre sum = % .8e\n", amp_gs_r, amp_gs_i, M_soliton_at_re);
    printf("XCEN = %.4e ; YCEN = %.4e ; ZCEN = % .4e\n", xcen, ycen, zcen);
    printf("=================================================\n");

    FILE *parameter_log = fopen("Record__Note", "w");
    fprintf(parameter_log, "Input Parameters:\n");
    fprintf(parameter_log, "R_RANGE = %.8e\n", r_range);
    fprintf(parameter_log, "BOX_RANGE = %.8e\n", box_range);
    fprintf(parameter_log, "R_EIGEN = %.8e\n", r_eigen);
    fprintf(parameter_log, "NBOX = %d\n", nbox);
    fprintf(parameter_log, "RSIZ = %d\n", rsiz);
    fprintf(parameter_log, "LSIZ = %d\n", Lsiz);
    fprintf(parameter_log, "NSIZ = %d\n", nsiz);
    fprintf(parameter_log, "LNOD = %d\n", lnod);
    fprintf(parameter_log, "A = %.8e\n", A);
    fprintf(parameter_log, "BETA = %.8e\n", beta);
    fprintf(parameter_log, "ESCAPE_E = %.8e\n", Ec);
    fprintf(parameter_log, "MU0 = %.8e\n", mu0);
    fprintf(parameter_log, "HALO_MASS = %.8e\n", halomass);
    fprintf(parameter_log, "PERTURB = %.8e\n", perturb_frac);
    fprintf(parameter_log, "OMEGA_M0 = %.8e\n", Omega_m0);
    fprintf(parameter_log, "SMALL_H0 = %.8e\n", h0);
    fprintf(parameter_log, "MASS_CORE_RATIO = %.8e\n", mass_core_ratio);
    fprintf(parameter_log, "NFW_C = %.8e\n", NFW_C);
    fprintf(parameter_log, "PHASE_BETWEEN_RI = %.8e\n", phase_between_RI);
    fprintf(parameter_log, "CRITERIA = %.8e\n", criteria);
    fprintf(parameter_log, "CONTRAST_FACTOR = %.8e\n", CF);
# ifdef OCTANT_DECOMPOSE
    fprintf(parameter_log, "OCTANT = %d\n", octant);
# endif
    fprintf(parameter_log, "DUMP_FLAG = %d\n", dump_flag);
    fprintf(parameter_log, "DUMP_INTERVAL = %d\n", dump_interval);
    fprintf(parameter_log, "RESTART_FLAG = %d\n", restart_flag);
    fprintf(parameter_log, "RESTART_ID = %d\n", restart_id);
    fprintf(parameter_log, "THREADS_PER_BLOCK_X = %d\n", tpB_x);
    fprintf(parameter_log, "THREADS_PER_BLOCK_Y = %d\n", tpB_y);
    fprintf(parameter_log, "THREADS_PER_BLOCK_Z = %d\n", tpB_z);
    fprintf(parameter_log, "BLOCKS_PER_GRID_X = %d\n", bpG_x);
    fprintf(parameter_log, "BLOCKS_PER_GRID_Y = %d\n", bpG_y);
    fprintf(parameter_log, "BLOCKS_PER_GRID_Z = %d\n", bpG_z);
    fprintf(parameter_log, "=================================================\n");
    fprintf(parameter_log, "Calculated Parameters:\n");
    fprintf(parameter_log, "ETA = %.8e\n", eta);
    fprintf(parameter_log, "CORE_R = %.8e\n", r_c);
    fprintf(parameter_log, "VIRIAL_R = %.8e ; POTENTIAL_VIRIAL = %.8e .\n", r_vir, phi_total_r(r_vir));
    fprintf(parameter_log, "EQUAL_R = %.8e\n", r_e);
    fprintf(parameter_log, "GSAMPR = %.8e ; GSAMP_I = %.8e ; squre sum = % .8e\n", amp_gs_r, amp_gs_i, M_soliton_at_re);
    fprintf(parameter_log, "XCEN = %.4e ; YCEN = %.4e ; ZCEN = % .4e\n", xcen, ycen, zcen);
    fprintf(parameter_log, "=================================================\n");
    fclose(parameter_log);

//    debug
//    printf("Virial radius is %.16e Mpc/h.\n", r_vir);
//    printf("The radius where rho_soliton and rho_NFW are equal is %.16e Mpc/h.\n", r_e);
//    printf("NFW Mass inside %.8e is %.16e .\n", r_vir, M_NFW_r(r_vir));
//    printf("%.16e\n", phi_total_r(4.*r_c));
//    printf("%.16e\n", phi_NFW_r(3.*r_c));
//    printf("%.8e\t%.8e\n", newton_g, eta);

//    adding calculation of mass of soliton inside a given radius
//    double M_core = M_soliton_r(r_c);
//    printf("Soliton mass inside %.8e is %.16e .\n", r_c, M_core);
//    printf("Soliton mass inside %.8e is %.16e .\n", r_e, M_soliton_at_re);
    return;
}
/*The authors of this work have released all rights to it and placed it
 *in the public domain_single_lidx_single_lidx under the Creative Commons CC0 1.0 waiver
 *(http://creativecommons.org/publicdomain_single_lidx_single_lidx/zero/1.0/).
 * 
 *THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 *EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 *MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 *IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 *CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 *TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 *SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 * 
 *Retrieved from: http://en.literateprograms.org/Box-Muller_transform_(C)?oldid=7011
 */
double rand_normal(double mean, double stddev)
{//Box muller method
    static double n2 = 0.0;
    static int n2_cached = 0;
    if (!n2_cached)
    {
        double x, y, r;
        do
        {
            x = 2.0*rand()/RAND_MAX - 1;
            y = 2.0*rand()/RAND_MAX - 1;

            r = x*x + y*y;
        }
        while (r == 0.0 || r > 1.0);
        {
            double d = sqrt(-2.0*log(r)/r);
            double n1 = x*d;
            n2 = y*d;
            double result = n1*stddev + mean;
            n2_cached = 1;
            return result;
        }
    }
    else
    {
        n2_cached = 0;
        return n2*stddev + mean;
    }
}
//double King(double energy,double A,double beta,double Ec)
//{	
//	double p=A*(exp(-beta*(energy-Ec))-1.);
//	if(p>=0.)
//		return p;
//	else
//		return 0.;
//}
//double Fermionic_King(double energy,double A,double beta,double Ec,double mu0)
//{
//	double p=A*(exp(-beta*(energy-Ec))-1.)/(1.+exp(-beta*(energy-Ec-mu0)));
//	if(p>=0.)
//		return p;
//	else
//		return 0.;
//}
//void Reset_GS_amp(double* amp_gs_r,double* amp_gs_i,double gs_eigenfun[])
//{
//	double mass_times_radius=8.56942249562e-07;
//	double* gs_rho;
//	double mass_radius=0.;
//	bool reset=true;
//
//	if(rank==0)
//	{
//		gs_rho=(double*)malloc(rsiz*sizeof(double));
//		for(int i=0;i<rsiz;i++)
//		{
//			gs_rho[i]=((*amp_gs_r)*(*amp_gs_r)+(*amp_gs_i)*(*amp_gs_i))*gs_eigenfun[i]*gs_eigenfun[i]/4./M_PI;
//			if(gs_rho[i]<=0.5*gs_rho[0]){
//				mass_radius=MassRadius(gs_rho,r,i);
//				printf("mass_radius=%e\n",mass_radius);//debug
//				if(reset){
//				(*amp_gs_r)=(*amp_gs_r)*sqrt(mass_times_radius/mass_radius);
//				(*amp_gs_i)=(*amp_gs_i)*sqrt(mass_times_radius/mass_radius);
//					i=-1;
//					reset=false;
//					continue;
//				}
//
//				break;
//			}
//				
//		}
//
//		
//		free(gs_rho);
//	}
//        MPI_Barrier(MPI_COMM_WORLD);
//	MPI_Bcast(amp_gs_r, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
//	MPI_Bcast(amp_gs_i, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
//	return;
//}//end Reset_GS_amp
//double MassRadius(double gs_rho[],double r[],int i)
//{
//	double mass_radius=0.;
//	double mass=0.;
//	
//	for(int ri=0;ri<=i;ri++)
//	{
//		mass+=4.*M_PI*r[ri]*r[ri]*dr*gs_rho[ri];
//	}
//	printf("i=%d,mass=%e,radius=%e\n",i,mass,r[i]);
//	mass_radius=mass*r[i];
//	
//	return mass_radius;
//}
//
//void set_2ndbox(void)
//{
//	strcpy(file_r,"32Cube_Center_lv6_Binary_Real");
//	strcpy(file_i,"32Cube_Center_lv6_Binary_Imag");
//
//	int ixcen = (int)rint(xcen / dx);
//    	int iycen = (int)rint(ycen / dy);
//    	int izcen = (int)rint(zcen / dz);
//
//	nx=nbox2*2;
//	ny=nbox2*2;
//	nz=nbox2*2;
//	
//	xcen=xcen-(ixcen-nbox2/2)*dx;
//	ycen=ycen-(iycen-nbox2/2)*dy;
//	zcen=zcen-(izcen-nbox2/2)*dz;
////	printf("xcen=%e ycen=%e zcen=%e\n",xcen,ycen,zcen);
//
//	dx=dx/2.;
//	dy=dy/2.;
//	dz=dz/2.;
//
//	return;
//}
