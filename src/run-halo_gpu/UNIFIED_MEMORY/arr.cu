#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include"cuapi.h"
#include"macros.h"
#include"main.h"
#include"arr.h"
#include"ylm.h"

__device__ int get_pos(int i, int j, int k, int nx, int ny)
{
    int pos = (i + nx * (j + ny * k));
    return pos;
}

__device__ void cal_dis_x(int i, float dx, float xcen, float *x)
{
    *x = i * dx - xcen;
}

__device__ void cal_dis_y(int j, float dy, float ycen, float *y)
{
    *y = j * dy - ycen;
}

__device__ void cal_dis_z(int k, float dz, float zcen, float *z)
{
    *z = k * dz - zcen;
}

__device__ void cal_dis_x_O(int i, int nx, int octant, float dx, float xcen, float *x)
{
    if((octant%4<=1))
        *x = (i+nx) * dx - xcen;
    else
        *x = i * dx - xcen;
}

__device__ void cal_dis_y_O(int j, int ny, int octant, float dy, float ycen, float *y)
{
    if((octant-1)%4<=1)
        *y = (j+ny) * dy - ycen;
    else
        *y = j * dy - ycen;
}

__device__ void cal_dis_z_O(int k, int nz, int octant, float dz, float zcen, float *z)
{
    if(octant<=4)
        *z = (k+nz) * dz - zcen;
    else
        *z = k * dz - zcen;
}

__device__ void cal_dis_r(float *x, float *y, float *z, float *r)
{
    *r = sqrtf((*x) * (*x) + (*y) * (*y) + (*z) * (*z));
}

void read_array(bool restart, int restart_id, char *filename_r, char *filename_i)
{
    FILE *f_temp[2];
    f_temp[0] = fopen(filename_r, "rb");
    f_temp[1] = fopen(filename_i, "rb");

#ifdef DEBUG
    if (f_temp[0]!=NULL)
        printf("Restart file for real part opened successfully!.\n");
    else
    {
        printf("Fail to open restart file for real part!.\n");
        exit(1);
    }
    if (f_temp[1]!=NULL)
        printf("Restart file for imaginary part opened successfully!.\n");
    else
    {
        printf("Fail to open restart file for imaginary part!.\n");
        exit(1);
    }
#endif

    if (restart)
    {
        int restart_id_check_r, nbox_check_r;
        int restart_id_check_i, nbox_check_i;
        fread(&restart_id_check_r, sizeof(int), 1, f_temp[0]);
        fread(&nbox_check_r, sizeof(int), 1, f_temp[0]);
        fread(&restart_id_check_i, sizeof(int), 1, f_temp[1]);
        fread(&nbox_check_i, sizeof(int), 1, f_temp[1]);

        restart_id_check_r ++;
        restart_id_check_i ++;
        if (restart_id_check_r!=restart_id||restart_id_check_i!=restart_id)
        {
            printf("RESTART_ID check failed while reading array (RESTART_ID=%d ; RESTART_ID_CHECK_R=%d ; RESTART_ID_CHECK_I=%d )! Exit!\n", restart_id, restart_id_check_r, restart_id_check_i);
            exit(1);
        }
        if (nbox_check_r!=nbox||nbox_check_i!=nbox)
        {
            printf("NBOX check failed while reading array (NBOX=%d ; NBOX_CHECK_R=%d ; NBOX_CHECK_I=%d )! Exit!\n", nbox, nbox_check_r, nbox_check_i);
            exit(1);
        }
    }
    fread(array_r_host, sizeof(float), nx*ny*nz, f_temp[0]);
    fread(array_i_host, sizeof(float), nx*ny*nz, f_temp[1]);

    fclose(f_temp[0]);
    fclose(f_temp[1]);
    return;
}
//

void write_array(bool dump, int *dump_id, char *filename_r, char *filename_i)
{
#ifdef OCTANT_DECOMPOSE
    sprintf(filename_r, "%s_octant%d", filename_r, octant);
    sprintf(filename_i, "%s_octant%d", filename_i, octant);
#endif

    if (dump)
    {
        sprintf(filename_r, "%s_DUMP_ID=%d", filename_r, *dump_id);
        sprintf(filename_i, "%s_DUMP_ID=%d", filename_i, *dump_id);
    }

    FILE *f_temp[2];
    f_temp[0] = fopen(filename_r, "wb");
    f_temp[1] = fopen(filename_i, "wb");

#ifdef DEBUG
    if (f_temp[0]!=NULL)
        printf("File for writing real part opened successfully!.\n");
    else
    {
        printf("Fail to open file for writing real part!.\n");
        exit(1);
    }
    if (f_temp[1]!=NULL)
        printf("File for writing imaginary part opened successfully!.\n");
    else
    {
        printf("Fail to open file for writing imaginary part!.\n");
        exit(1);
    }
#endif

    if (dump)
    {
        fwrite(dump_id, sizeof(int), 1, f_temp[0]);
        fwrite(&nbox, sizeof(int), 1, f_temp[0]);
        fwrite(dump_id, sizeof(int), 1, f_temp[1]);
        fwrite(&nbox, sizeof(int), 1, f_temp[1]);
    }

    fwrite(array_r_host, sizeof(float), nx*ny*nz, f_temp[0]);
    fwrite(array_i_host, sizeof(float), nx*ny*nz, f_temp[1]);

    fclose(f_temp[0]);
    fclose(f_temp[1]);
    return;
}

void array_init()
{
	array_r_host = (float*)malloc(N_site*sizeof(float));
	array_i_host = (float*)malloc(N_site*sizeof(float));
        memset(array_r_host, 0, N_site*sizeof(float));
        memset(array_i_host, 0, N_site*sizeof(float));
	return;
}

void array_fin()
{
    free(array_r_host);
    free(array_i_host);
}

void array_add_ylm(int l_max,int l_init,bool firstadd)
{
    bool timing_flag;
    bool cuda_error_flag;
    hipEvent_t start, stop;
    float total_execute, total_cal_and_expand;
    float *array_r_device, *array_i_device;
    float *array_r_l, *array_i_l;

    cuda_error_flag = CUDA_CHECK_ERROR(hipEventCreate(&start));
    cuda_error_flag = CUDA_CHECK_ERROR(hipEventCreate(&stop));
    cuda_error_flag = CUDA_CHECK_ERROR(hipEventRecord(start));
#ifdef OCTANT_DECOMPOSE
    printf( "Working on Octant [%d/8] -- Lidx [%3d/%3d] ...\n", octant_global, Lidx_global, lnod-1);
#else
    printf( "Working on Lidx [%3d/%3d] ...\n", Lidx_global, lnod-1);
#endif

    timing_flag = true;
    if (timing_flag)
        total_cal_and_expand = 0.;

    cuda_error_flag = CUDA_CHECK_ERROR(hipMalloc((void **)&array_r_device, N_site*sizeof(float)));
    if (!cuda_error_flag)
    {
        printf("Memory allocating array_r_device failed!\n");
	exit(1);
    }
#ifdef DEBUG
    else
        printf("Memory allocating array_r_device succeed!\n");
#endif
    cuda_error_flag = CUDA_CHECK_ERROR(hipMalloc((void **)&array_i_device, N_site*sizeof(float)));
    if (!cuda_error_flag)
    {
        printf("Memory allocating array_i_device failed!\n");
	exit(1);
    }
#ifdef DEBUG
    else
        printf("Memory allocating array_i_device succeed!\n");
    printf("CUDA malloc completed...\n");
#endif
    cuda_error_flag = CUDA_CHECK_ERROR(hipMemset(array_r_device, 0, N_site*sizeof(float)));
    if (!cuda_error_flag)
    {
        printf("Memory setting array_r failed!\n");
	exit(1);
    }
#ifdef DEBUG
    else
        printf("Memory setting array_r succeed!\n");
#endif
    cuda_error_flag = CUDA_CHECK_ERROR(hipMemset(array_i_device, 0, N_site*sizeof(float)));
    if (!cuda_error_flag)
    {
        printf("Memory setting array_i failed!\n");
	exit(1);
    }
#ifdef DEBUG
    else
        printf("Memory setting array_i succeed!\n");
#endif
//    array_r_l = (float*)malloc(N_site*sizeof(float));
//    array_i_l = (float*)malloc(N_site*sizeof(float));
    cuda_error_flag = CUDA_CHECK_ERROR(hipHostMalloc((void**)&array_r_l, N_site*sizeof(float)));
    cuda_error_flag = CUDA_CHECK_ERROR(hipHostMalloc((void**)&array_i_l, N_site*sizeof(float)));

    do_expand_ylm(array_r_device, array_i_device, &total_cal_and_expand, l_max,l_init, timing_flag);
    
    cuda_error_flag = CUDA_CHECK_ERROR(hipMemcpy(array_r_l, array_r_device, N_site*sizeof(float), hipMemcpyDeviceToHost));
    if (!cuda_error_flag)
        printf("Copying array_r failed!\n");
#ifdef DEBUG
    else
        printf("Copying array_r succeed!\n");
#endif
        
    cuda_error_flag = CUDA_CHECK_ERROR(hipMemcpy(array_i_l, array_i_device, N_site*sizeof(float), hipMemcpyDeviceToHost));
    if (!cuda_error_flag)
        printf("Copying array_i failed!\n");
#ifdef DEBUG
    else
        printf("Copying array_i succeed!\n");
#endif
    
    cuda_error_flag = CUDA_CHECK_ERROR(hipFree(array_r_device));
    cuda_error_flag = CUDA_CHECK_ERROR(hipFree(array_i_device));

    for (int site=0; site<N_site; site++)
    {
        array_r_host[site] += array_r_l[site];
        array_i_host[site] += array_i_l[site];
    }
//    free(array_r_l);
//    free(array_i_l);
    cuda_error_flag = hipHostFree(array_r_l);
    cuda_error_flag = hipHostFree(array_i_l);

    cuda_error_flag = CUDA_CHECK_ERROR(hipEventRecord(stop));
    cuda_error_flag = CUDA_CHECK_ERROR(hipEventSynchronize(stop));
    cuda_error_flag = CUDA_CHECK_ERROR(hipEventElapsedTime(&total_execute, start, stop));

    printf("Total time for expanding-calculating ylm of l_init=%d to l_max=%d over all k takes %.4e s.\n", l_init, l_max, total_cal_and_expand);
    printf("Total time for adding contribution of l_init=%d to l_max=%d over all k takes %.4e s.\n", l_init, l_max, total_execute/1000.);
    fflush(stdout);

    return;
}

void cal_r_max(void)
{
    double x_max;
    double y_max;
    double z_max;

    x_max = min( xcen, (nx-1)*dx - xcen);
    y_max = min( ycen, (ny-1)*dy - ycen);
    z_max = min( zcen, (nz-1)*dz - zcen);

    r_max = nx*dx + ny*dy + nz*dz;
    r_max = min(r_max, x_max);
    r_max = min(r_max, y_max);
    r_max = min(r_max, z_max);
    r_max *= r_fact;
    return;
}

void cal_r_max_O(void)
{
    dx=dx/2.;
    dy=dy/2.;
    dz=dz/2.;

    double x_max;
    double y_max;
    double z_max;

    x_max = min( xcen, (2*nx-1)*dx - xcen);
    y_max = min( ycen, (2*ny-1)*dy - ycen);
    z_max = min( zcen, (2*nz-1)*dz - zcen);

    r_max = nx*dx + ny*dy + nz*dz;
    r_max = min(r_max, x_max);
    r_max = min(r_max, y_max);
    r_max = min(r_max, z_max);
    r_max *= r_fact;
//	printf("r_max=%e\n",r_max);//debug
    return;
}
