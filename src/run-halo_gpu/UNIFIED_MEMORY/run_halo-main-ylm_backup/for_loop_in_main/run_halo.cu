/* move extern int Lidx_global and extern int octant_global to arr.h  2020.12.29*/
/* fix typo in cal_dis_O() 2020.12.30 */
/* add timing in array_add_ylm() 2020.12.31 */
/* add r_max debug in array_add_ylm() 2020.12.31 */
/* add auto renaming output file in write_array() 2021.01.03 */
/* add iter_test for testing the code serval times 2021.01.07*/
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include"cuapi.h"
#include"macros.h"
#include"text_io.h"
#include"main.h"
#include"arr.h"
#include"ylm.h"
#define RESTART_FILENAME_R "RESTART_R"
#define RESTART_FILENAME_I "RESTART_I"

void run_single_lidx(void);

void run(void)
{
    run_single_lidx();
}

void run_single_lidx(void)
{
    float time_temp;
    char buff_r[128], buff_i[128];
    
//    solve_init();
//    gen_r(r_1,r);
	
    int l_max=0;
    int l_init=0;
    
// add data dump and restart 2021.01.04
    hipEvent_t eigen_copy_start, eigen_copy_stop, amplitude_copy_start, amplitude_copy_stop, lidx_start, lidx_stop;
    hipSetDevice(0);

    hipEventCreate(&eigen_copy_start);
    hipEventCreate(&eigen_copy_stop);
    hipEventCreate(&amplitude_copy_start);
    hipEventCreate(&amplitude_copy_stop);
    hipEventCreate(&lidx_start);
    hipEventCreate(&lidx_stop);

    hipEventRecord(lidx_start,0);
    hipEventRecord(eigen_copy_start,0);
    
    /*** load eigenstates and broadcast to every nodes***/
    load_eigenstates(&l_init,&l_max,"egn",Lidx_global);
#ifdef DEBUG
    printf("Eigen states loading completed!\n");
#endif
    hipEventRecord(eigen_copy_stop,0);
    hipEventSynchronize(eigen_copy_stop);
    hipEventElapsedTime(&time_temp, eigen_copy_start, eigen_copy_stop);
    printf("Total time for copying eigen states from l_init=%d to l_max=%d is %.4e s.\n", l_init, l_max, time_temp/1000.);
//  ignoring all operations if l_max<l_init 2020.12.29
    if (l_max<l_init)
    {
        printf("l_init = %d @ Lidx = %d will be ignored since l_max < l_init !\n", l_init, Lidx_global);
        return;
    }
#ifdef DEBUG
    printf("Start kernel preparation...\n");
#endif
    /*** generate upper bound ***/
//        gen_ubound(rv[0][0] * 0., l_max,l_init);
    ylm_init(l_init,l_max);
#ifdef DEBUG
    printf("Memory allocation for spherical harmonics completed.\n");
#endif

    hipEventRecord(amplitude_copy_start,0);
    /*** read amplitudes and Bcast ***/
    load_ylm(l_init,l_max,"amp_only",Lidx_global);
    /*** generate density in simulation box ***/
#ifdef DEBUG
    printf("Amplitudes of spherical harmonics loading completed.\n");
#endif
    hipEventRecord(amplitude_copy_stop,0);
    hipEventSynchronize(amplitude_copy_stop);
    hipEventElapsedTime(&time_temp, amplitude_copy_start, amplitude_copy_stop);
    printf("Total time for copying  amplitudes from l_init=%d to l_max=%d is %.4e s.\n", l_init, l_max, time_temp/1000.);

    array_add_ylm(l_max,l_init,true);

    hipDeviceSynchronize();
    ylm_fin(l_init,l_max);
    free_egn(l_max,l_init);

    hipEventRecord(lidx_stop,0);
    hipEventSynchronize(lidx_stop);
    hipEventElapsedTime(&time_temp, lidx_start, lidx_stop);
    printf("Total time for running through l_init=%d to l_max=%d takes %.4e s.\n",l_init, l_max, time_temp/1000.);

    hipDeviceReset();

    return;
}
